#include "hip/hip_runtime.h"
#include "model.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

structParameters ParametersInitialize(){
    structParameters params;
    params.micDiffusion = 3*24*6.6*pow(10,-5);
    params.antibodyDiffusion = 9.6*24*6.6*pow(10,-4);
    params.cDcDiffusion = 9.6*24*6.6*pow(10,-6);
    params.aDcDiffusion = 9.6*24*6.6*pow(10,-5);
    params.tCytoDiffusion = 50.6*24*6.6*pow(10,-5);
    params.chi = 0.03;
    
    params.muCDc = 60*24*3*pow(10,-5);
    params.muMic = 60*24*3*pow(10,-6);
    params.rM = 60*24*6*pow(10,-7);
    params.rT = 0.001;
    params.lambAntMic = 5.702*pow(10,-3);
    params.bD = 0.001;
    
    params.gammaD = 0.1;
    params.gammaAntibody = 0.3;
    params.gammaT = 0.1;

    params.avgT = 37;
    params.avgDc = 33;
    params.avgMic = 350;
    params.avgOdc = 400;

    params.cMic = 0.1;
    params.cCDc = 1;
    params.cADc = 1;
    params.cDl = 0.1;
    params.cF = 0.1;
    params.alphaTHelper = 0.1;
    params.alphaTCytotoxic = 0.1;
    params.alphaB = 0.1;
    params.alphaP = 1;
    params.bTHelper = 0.17;
    params.bTCytotoxic = 0.001;
    params.bRho = 0.6;
    params.bRhoB = 3.02;
    params.bRhoP = 1.02;
    params.rhoTHelper = 2;
    params.rhoTCytotoxic = 2;
    params.rhoB = 11;
    params.rhoP = 3;
    params.rhoAntibody = 5.1*pow(10,-2);
    params.stableTHelper = 70;
    params.stableTCytotoxic = 40;
    params.stableB = 25;
    params.stableP = 2.5;
    params.V_LN = 40;
    params.V_BV = 0;
    params.V_PV = 0;

    return params;
}

void WriteTime(float ExecTime){
    FILE *fileTime;
    fileTime = fopen("./ExecsTimes.txt", "a");
    if(fileTime != NULL){
    fprintf(fileTime, "%f\n", ExecTime);
    fclose(fileTime);
    }else{
        printf("Error execution time file\n");
        exit(0);
    }
}

void clearPhgTxt(){
    system("find ./result/ -name '*.png' -type f -delete");
    system("find ./result/ -name '*.txt' -type f -delete");
    system("mkdir result");
    system("mkdir result/matrix");
    system("mkdir result/odc");
    system("mkdir result/mic");
    system("mkdir result/tke");
    system("mkdir result/ant");
    system("mkdir result/da");
    system("mkdir result/dc");
}

int main(){
    clearPhgTxt();
    float ht = 0.0002, hx = 0.5;
    int numFigs = 28, numPointsLN = 1000, time = 28, space = 20, numStepsLN = 1, saveFigs = 1;
    structParameters parameters = ParametersInitialize();
    structModel model = ModelInitialize(parameters, ht, hx, time, space, numFigs, numPointsLN, numStepsLN, saveFigs);
    RunModel(&model);
    return 0;
}
