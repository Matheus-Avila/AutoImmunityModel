#include "hip/hip_runtime.h"
#include "model.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

structParameters ReadParameters(){
    FILE *file;
    file = fopen("./sensitivity_analysis/SA_parameters.txt","r");
    char lineRead[25];
    structParameters params;
    float fileParameters[34];
    int fileIter = 0;
    while (fgets(lineRead, sizeof(lineRead), file) != NULL){
        // char* valParam = strtok(lineRead, "\n");
        fileParameters[fileIter] = atof(lineRead);
        fileIter++;
    }
    fclose(file);
    //Escrever o vetor de parametros na estrutura parametros
    params.chi = fileParameters[0];
    params.micDiffusion = fileParameters[1];
    params.cDcDiffusion = fileParameters[2];
    params.aDcDiffusion = fileParameters[3];
    params.tCytoDiffusion = fileParameters[4];
    params.antibodyDiffusion = fileParameters[5];
    
    params.muMic = fileParameters[6];
    params.rM = fileParameters[7];
    params.lambAntMic = fileParameters[8];
    params.bD = fileParameters[9];
    params.rT = fileParameters[10];
    params.muCDc = fileParameters[11];
    
    params.gammaD = fileParameters[12];
    params.gammaAntibody = fileParameters[13];
    params.gammaT = fileParameters[14];

    params.alphaTHelper = fileParameters[15];
    params.alphaTCytotoxic = fileParameters[16];
    params.alphaB = fileParameters[17];
    params.alphaP = fileParameters[18];
    params.cMic = fileParameters[19];
    params.cCDc = fileParameters[20];
    params.cADc = fileParameters[21];
    params.cDl = fileParameters[22];
    params.cF = fileParameters[23];
    params.bTHelper = fileParameters[24];
    params.bTCytotoxic = fileParameters[25];
    params.bRho = fileParameters[26];
    params.bRhoB = fileParameters[27];
    params.bRhoP = fileParameters[28];
    params.rhoTHelper = fileParameters[29];
    params.rhoTCytotoxic = fileParameters[30];
    params.rhoB = fileParameters[31];
    params.rhoP = fileParameters[32];
    params.rhoAntibody = fileParameters[33];


    params.avgT = 37;
    params.avgDc = 33;
    params.avgMic = 350;
    params.avgOdc = 400;
    params.estableTHelper = 84;
    params.estableTCytotoxic = 40;
    params.estableB = 25;
    params.estableP = 2.5;
    params.V_LN = 40;
    params.V_BV = 0;
    params.V_PV = 0;
    return params;
}


structParameters ParametersInitialize(){
    structParameters params;
    params.micDiffusion = 0.015206;
    params.antibodyDiffusion = 0.15206;
    params.cDcDiffusion = 0.015206;
    params.aDcDiffusion = 0.015206;
    params.tCytoDiffusion = 0.015206;
    params.chi = 0.03;
    
    params.muCDc = 60*24*3*pow(10,-5);
    params.muMic = 60*24*3*pow(10,-6);
    params.rM = 60*24*6*pow(10,-7);
    params.rT = 0.001;
    params.lambAntMic = 5.702*pow(10,-3);
    params.bD = 0.001;
    
    params.gammaD = 0.1;
    params.gammaAntibody = 0.3;
    params.gammaT = 0.1;

    params.avgT = 37;
    params.avgDc = 33;
    params.avgMic = 350;
    params.avgOdc = 400;

    params.cMic = 0.1;
    params.cCDc = 1;
    params.cADc = 1;
    params.cDl = 0.1;
    params.cF = 0.1;
    params.alphaTHelper = 0.1;
    params.alphaTCytotoxic = 0.1;
    params.alphaB = 0.1;
    params.alphaP = 1;
    params.bTHelper = 0.17;
    params.bTCytotoxic = 0.001;
    params.bRho = 0.6;
    params.bRhoB = 3.02;
    params.bRhoP = 1.02;
    params.rhoTHelper = 2;
    params.rhoTCytotoxic = 2;
    params.rhoB = 11;
    params.rhoP = 3;
    params.rhoAntibody = 5.1*pow(10,-2);
    params.estableTHelper = 70;
    params.estableTCytotoxic = 40;
    params.estableB = 25;
    params.estableP = 2.5;
    params.V_LN = 40;
    params.V_BV = 0;
    params.V_PV = 0;

    return params;
}

void WriteTime(float ExecTime){
    FILE *fileTime;
    fileTime = fopen("./ExecsTimes.txt", "a");
    if(fileTime != NULL){
    fprintf(fileTime, "%f\n", ExecTime);
    fclose(fileTime);
    }else{
        printf("Error execution time file\n");
        exit(0);
    }
}

void clearPhgTxt(){
    system("find ./result/ -name '*.png' -type f -delete");
    system("find ./result/ -name '*.txt' -type f -delete");
    system("mkdir result");
    system("mkdir result/matrix");
    system("mkdir result/odc");
    system("mkdir result/mic");
    system("mkdir result/tke");
    system("mkdir result/ant");
    system("mkdir result/da");
    system("mkdir result/dc");
}

int main(int argc, char** argv){
    int perform_analysis = 0;
    structParameters parameters;
    if(argc > 1) {
       perform_analysis = strtol(argv[1], NULL, 10);
    }

    clearPhgTxt();
    if(perform_analysis <= 0) {
        parameters = ParametersInitialize();
    }
    else {
        parameters = ReadParameters();
    }
    float ht = 0.0002, hx = 0.05;
    int numFigs = 28, numPointsLN = 1000, time = 28, space = 20, numStepsLN = 100, saveFigs = 0;
    
    structModel model = ModelInitialize(parameters, ht, hx, time, space, numFigs, numPointsLN, numStepsLN, saveFigs);
    RunModel(&model);
    return 0;
}