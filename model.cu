#include "hip/hip_runtime.h"
#include "model.h"
#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

void InitialConditionTissueMicroglia(structModel* model){
    for(int k = 0; k < model->xSize*model->xSize; k++){
        int i = (int)k/model->xSize;
        int j = k%model->xSize;
        if(pow((i-(int)(model->xSize/2)),2) + pow((j-(int)(model->xSize/2)),2) < 5 / (model->hx * model->hx)){
            model->microglia[0][k] = (float)model->parametersModel.avgMic/3;
        }
    }
}

void InitialConditionLymphNode(structModel *model, float dendriticLN, float thelperLN, float tcytotoxicLN, float bcellLN, float plasmacellLN, float antibodyLN)
{
    model->dendriticLymphNode[0] = dendriticLN;
    model->tHelperLymphNode[0] = thelperLN;
    model->tCytotoxicLymphNode[0] = tcytotoxicLN;
    model->bCellLymphNode[0] = bcellLN;
    model->plasmaCellLymphNode[0] = plasmacellLN;
    model->antibodyLymphNode[0] = antibodyLN;
}

int VerifyCFL(structParameters parametersModel, float ht, float hx){
    if(parametersModel.micDiffusion*ht/(hx*hx) < 0.25 && parametersModel.cDcDiffusion*ht/(hx*hx) < 0.25 && parametersModel.aDcDiffusion*ht/(hx*hx) < 0.25 && parametersModel.tCytoDiffusion*ht/(hx*hx) < 0.25 && parametersModel.chi*ht/hx < 0.5)
        return 1;
    return 0;
}

void WritePopulation(structModel model, float *population, char *fileName, char *bufferTime)
{
    FILE *file;
    file = fopen(fileName, "w");
    int k = 0;
    while (k < model.xSize * model.xSize)
    {
        int i = k;
        while (i < k + model.xSize)
        {
            fprintf(file, "%f ", population[i]);
            i++;
        }
        fprintf(file, "\n");
        k += model.xSize;
    }
    fclose(file);
}

void WritePopulationLymphNode(structModel model, float *population, char *fileName)
{
    FILE *file;
    file = fopen(fileName, "w");
    for (int i = 0; i < model.numPointsLN; i++)
    {
        fprintf(file, "%f\n", population[i]);
    }
    fclose(file);
}

void WriteLymphNodeFiles(structModel model, float *dendritic, float *tHelper, float *tCytotoxic, float *bCell, float *plasmaCell, float *antibody)
{
    WritePopulationLymphNode(model, dendritic, "./result/dendritic.txt");
    WritePopulationLymphNode(model, tHelper, "./result/tHelper.txt");
    WritePopulationLymphNode(model, tCytotoxic, "./result/tCyto.txt");
    WritePopulationLymphNode(model, bCell, "./result/bCell.txt");
    WritePopulationLymphNode(model, plasmaCell, "./result/plasmaCell.txt");
    WritePopulationLymphNode(model, antibody, "./result/antibody.txt");

    char buffer[10];
    char command[40] = {};
    strcat(command, "python3 plotLymphNode.py ");
    snprintf(buffer, sizeof(buffer), "%d", model.tFinal);
    strcat(command, buffer);
    strcat(command, " ");
    snprintf(buffer, sizeof(buffer), "%f", (model.tSize / model.numPointsLN) * model.ht);
    strcat(command, buffer);
    system(command);
}

void WriteFiles(structModel model, float *oligodendrocyte, float *microglia, float *tCytotoxic, float *antibody, float *conventionalDC, float *activatedDC, float time)
{
    char buffer[10];
    float day = time * model.ht;

    snprintf(buffer, sizeof(buffer), "%.1f", day);

    char pathOligodendrocytes[70] = "./result/matrix/oligo";
    strcat(pathOligodendrocytes, buffer);
    strcat(pathOligodendrocytes, ".txt");
    WritePopulation(model, oligodendrocyte, pathOligodendrocytes, buffer);

    char pathMicroglia[70] = "./result/matrix/microglia";
    strcat(pathMicroglia, buffer);
    strcat(pathMicroglia, ".txt");
    WritePopulation(model, microglia, pathMicroglia, buffer);

    char pathTCyto[70] = "./result/matrix/tCyto";
    strcat(pathTCyto, buffer);
    strcat(pathTCyto, ".txt");
    WritePopulation(model, tCytotoxic, pathTCyto, buffer);

    char pathAntibody[70] = "./result/matrix/antibody";
    strcat(pathAntibody, buffer);
    strcat(pathAntibody, ".txt");
    WritePopulation(model, antibody, pathAntibody, buffer);

    char pathConventionalDC[70] = "./result/matrix/conventionalDC";
    strcat(pathConventionalDC, buffer);
    strcat(pathConventionalDC, ".txt");
    WritePopulation(model, conventionalDC, pathConventionalDC, buffer);

    char pathActivatedDC[70] = "./result/matrix/activatedDC";
    strcat(pathActivatedDC, buffer);
    strcat(pathActivatedDC, ".txt");
    WritePopulation(model, activatedDC, pathActivatedDC, buffer);
}

void PlotResults(structModel model)
{
    char buffer[10];
    char command[70] = {};
    strcat(command, "python3 plotMatrices.py ");
    snprintf(buffer, sizeof(buffer), "%d", model.xFinal);
    strcat(command, buffer);
    strcat(command, " ");
    snprintf(buffer, sizeof(buffer), "%f", model.hx);
    strcat(command, buffer);
    strcat(command, " ");
    snprintf(buffer, sizeof(buffer), "%d", model.tFinal);
    strcat(command, buffer);
    strcat(command, " ");
    snprintf(buffer, sizeof(buffer), "%d", model.intervalFigures);
    strcat(command, buffer);
    system(command);
}

__device__ void AdvectionTerm(float populationPoint, float avgValue, float *result)
{
    *result = populationPoint / (populationPoint + avgValue);
}

__device__ void UpDownWind(float frontPoint, float rearPoint, float avgValue, float *result)
{
    float resultF;
    AdvectionTerm(frontPoint, avgValue, result);
    AdvectionTerm(rearPoint, avgValue, &resultF);
    *result = *result - resultF;
}

__device__ void CalculateChemottaxis(float hx, float frontJPoint, float rearJPoint, float frontIPoint, float rearIPoint, float ijPoint,
                                     float avgValue, float gradientOdcI, float gradientOdcJ, float *result)
{
    float gradientPopulationI, gradientPopulationJ;
    if (gradientOdcI < 0)
    {
        UpDownWind(frontIPoint, ijPoint, avgValue, &gradientPopulationI);
        gradientPopulationI = gradientPopulationI / (float)hx;
    }
    else
    {
        UpDownWind(ijPoint, rearIPoint, avgValue, &gradientPopulationI);
        gradientPopulationI = gradientPopulationI / (float)hx;
    }
    if (gradientOdcJ < 0)
    {
        UpDownWind(frontJPoint, ijPoint, avgValue, &gradientPopulationJ);
        gradientPopulationJ = gradientPopulationJ / (float)hx;
    }
    else
    {
        UpDownWind(ijPoint, rearJPoint, avgValue, &gradientPopulationJ);
        gradientPopulationJ = gradientPopulationJ / (float)hx;
    }

    *result = gradientOdcI * gradientPopulationI + gradientOdcJ * gradientPopulationJ;
}

__device__ void CalculateDiffusion(float hx, float frontJPoint, float rearJPoint, float frontIPoint, float rearIPoint, float ijPoint, float *result)
{
    *result = (float)(frontIPoint + frontJPoint - 4 * ijPoint + rearIPoint + rearJPoint) / (float)(hx * hx);
}

__device__ void fFunc(float valuePopulation, float avgPopulation, float *result)
{
    *result = valuePopulation * valuePopulation / (float)(valuePopulation + avgPopulation);
}

void WriteBVPV(structModel *model, float *thetaBV, float *thetaPV)
{
    FILE *fileBV;
    fileBV = fopen("./result/bv.txt", "w");
    FILE *filePV;
    filePV = fopen("./result/pv.txt", "w");
    for (int k = 0; k < model->xSize * model->xSize; k++)
    {
        fprintf(fileBV, "%f ", thetaBV[k]);
        fprintf(filePV, "%f ", thetaPV[k]);
        if (k % model->xSize == 0 && k != 0)
        {
            fprintf(fileBV, "\n");
            fprintf(filePV, "\n");
        }
    }
    fclose(fileBV);
    fclose(filePV);
    char buffer[10];
    char command[70] = {};
    strcat(command, "python3 plotBVPV.py ");
    snprintf(buffer, sizeof(buffer), "%d", model->xFinal);
    strcat(command, buffer);
    strcat(command, " ");
    snprintf(buffer, sizeof(buffer), "%f", model->hx);
    strcat(command, buffer);
    // system(command);
}

void DefineBVPV(structModel *model)
{
    int randomVal;
    for (int k = 0; k < model->xSize * model->xSize; k++)
    {
        int j = k % model->xSize;
        randomVal = rand() % 100;
        if (randomVal < 10)
        {
            model->parametersModel.V_BV++;
            model->parametersModel.V_PV++;
            model->thetaBV[k] = 1;
            if (j != model->xSize - 1)
                model->thetaPV[k + 1] = 1;
            else
                model->thetaPV[k - model->xSize + 1] = 1;
        }
    }
    model->parametersModel.V_BV = 160;//model->parametersModel.V_BV * model->hx * model->hx;
    model->parametersModel.V_PV = 160;//model->parametersModel.V_PV * model->hx * model->hx;
    WriteBVPV(model, model->thetaBV, model->thetaPV);
}

structModel ModelInitialize(structParameters params, float ht, float hx, float time, float space, int numFigs, int numPointsLN)
{
    structModel model;
    srand(2);

    // Pegar os valores pelos parametros
    model.parametersModel = params;
    model.numFigs = numFigs;
    model.numPointsLN = numPointsLN;
    model.ht = ht;
    model.hx = hx;
    model.tFinal = time;
    model.xFinal = space;
    model.tSize = (int)(time / ht);
    model.xSize = (int)(space / hx);
    model.intervalFigures = (int)model.tSize / numFigs;

    // inicializar dinamicamente todos os vetores do tecido
    model.microglia = (float **)calloc(BUFFER, sizeof(float *));
    model.oligodendrocyte = (float **)calloc(BUFFER, sizeof(float *));
    model.tCytotoxic = (float **)calloc(BUFFER, sizeof(float *));
    model.antibody = (float **)calloc(BUFFER, sizeof(float *));
    model.conventionalDc = (float **)calloc(BUFFER, sizeof(float *));
    model.activatedDc = (float **)calloc(BUFFER, sizeof(float *));
    for (int index = 0; index < BUFFER; index++)
    {
        model.microglia[index] = (float *)calloc(model.xSize * model.xSize, sizeof(float));
        model.oligodendrocyte[index] = (float *)calloc(model.xSize * model.xSize, sizeof(float));
        model.tCytotoxic[index] = (float *)calloc(model.xSize * model.xSize, sizeof(float));
        model.antibody[index] = (float *)calloc(model.xSize * model.xSize, sizeof(float));
        model.conventionalDc[index] = (float *)calloc(model.xSize * model.xSize, sizeof(float));
        model.activatedDc[index] = (float *)calloc(model.xSize * model.xSize, sizeof(float));
    }

    model.activatedDCTissueVessels = 0;
    model.tCytotoxicTissueVessels = 0;
    model.antibodyTissueVessels = 0;

    // definir BV e PV
    model.thetaPV = (float *)calloc(model.xSize * model.xSize, sizeof(float));
    model.thetaBV = (float *)calloc(model.xSize * model.xSize, sizeof(float));
    DefineBVPV(&model);
    // definir lymph node
    model.dendriticLymphNodeSavedPoints = (float *)calloc(model.numPointsLN, sizeof(float));
    model.tCytotoxicLymphNodeSavedPoints = (float *)calloc(model.numPointsLN, sizeof(float));
    model.tHelperLymphNodeSavedPoints = (float *)calloc(model.numPointsLN, sizeof(float));
    model.antibodyLymphNodeSavedPoints = (float *)calloc(model.numPointsLN, sizeof(float));
    model.bCellLymphNodeSavedPoints = (float *)calloc(model.numPointsLN, sizeof(float));
    model.plasmaCellLymphNodeSavedPoints = (float *)calloc(model.numPointsLN, sizeof(float));

    model.dendriticLymphNode = (float *)calloc(2, sizeof(float));
    model.tCytotoxicLymphNode = (float *)calloc(2, sizeof(float));
    model.tHelperLymphNode = (float *)calloc(2, sizeof(float));
    model.antibodyLymphNode = (float *)calloc(2, sizeof(float));
    model.bCellLymphNode = (float *)calloc(2, sizeof(float));
    model.plasmaCellLymphNode = (float *)calloc(2, sizeof(float));
    float dendriticLN = 0.0, thelperLN = 0.0, tcytotoxicLN = 0.0, bcellLN = 0.0, plasmacellLN = 0.0, antibodyLN = 0.0;
    InitialConditionLymphNode(&model, dendriticLN, thelperLN, tcytotoxicLN, bcellLN, plasmacellLN, antibodyLN);
    InitialConditionTissueMicroglia(&model);
    return model;
}

void verifyValues(structModel model, float value, int time, char* populationName){
    if(value < 0 ||  isnanf(value)){
        printf("Error: %s = (%f) :: time = %f\n", populationName, value, time*model.ht);
        exit(0);
    }
}

void verifyDerivate(structModel model, float value, int time, char* populationName){
    if(isnanf(value)){
        printf("Error: %s = (%f) :: time = %f\n", populationName, value, time*model.ht);
        exit(0);
    }
}

float *EquationsLymphNode(structModel model, float *populationLN, int stepPos)
{
    float *result = (float *)malloc(sizeof(float) * 6);

    float dcLN = populationLN[0];
    float tCytoLN = populationLN[1];
    float tHelperLN = populationLN[2];
    float bCellLN = populationLN[3];
    float plasmaCellLN = populationLN[4];
    float antibodyLN = populationLN[5];

    // Describe equations

    //Dendritic cell
    float activatedDcMigration = model.parametersModel.gammaD * (model.activatedDCTissueVessels - dcLN) * (float)(model.parametersModel.V_PV/model.parametersModel.V_LN);
    float activatedDcClearance = model.parametersModel.cDl * dcLN;
    result[0] = activatedDcMigration - activatedDcClearance;

    //T Cytotoxic
    float tCytoActivation = model.parametersModel.bTCytotoxic * (model.parametersModel.rhoTCytotoxic*tCytoLN*dcLN - tCytoLN*dcLN);
    float tCytoHomeostasis = model.parametersModel.alphaTCytotoxic * (model.parametersModel.estableTCytotoxic - tCytoLN);
    float tCytoMigration = model.parametersModel.gammaT * (tCytoLN - model.tCytotoxicTissueVessels) * (float)(model.parametersModel.V_BV/model.parametersModel.V_LN);
    result[1] = tCytoActivation + tCytoHomeostasis - tCytoMigration;

    //T Helper
    float tHelperActivation = model.parametersModel.bTHelper * (model.parametersModel.rhoTHelper * tHelperLN * dcLN - tHelperLN * dcLN);
    float tHelperHomeostasis = model.parametersModel.alphaTHelper * (model.parametersModel.estableTHelper - tHelperLN);
    float tHelperDispendure = model.parametersModel.bRho * dcLN * tHelperLN * bCellLN;
    result[2] = tHelperActivation + tHelperHomeostasis - tHelperDispendure;

    //B Cell
    float bCellActivation = model.parametersModel.bRhoB * (model.parametersModel.rhoB * tHelperLN * dcLN - tHelperLN * dcLN * bCellLN);
    float bcellHomeostasis = model.parametersModel.alphaB * (model.parametersModel.estableB - bCellLN);
    result[3] = bcellHomeostasis + bCellActivation;

    //Plasma Cells
    float plasmaActivation = model.parametersModel.bRhoP * (model.parametersModel.rhoP * tHelperLN * dcLN * bCellLN);
    float plasmaHomeostasis = model.parametersModel.alphaP * (model.parametersModel.estableP - plasmaCellLN);
    result[4] = plasmaHomeostasis + plasmaActivation;

    //Antibody
    float antibodyProduction = model.parametersModel.rhoAntibody * plasmaCellLN;
    float antibodyDecayment = model.parametersModel.cF * antibodyLN;
    float antibodyMigration = model.parametersModel.gammaAntibody * (antibodyLN - model.antibodyTissueVessels) * (float)(model.parametersModel.V_BV/model.parametersModel.V_LN);
    result[5] = antibodyProduction - antibodyMigration - antibodyDecayment;

    return result;
}


void SolverLymphNode(structModel *model, int stepPos)
{
    float populationLN[6];
    int stepKMinus = (stepPos - 1) % 2;
    int stepKPlus = (stepPos) % 2;
    populationLN[0] = model->dendriticLymphNode[stepKMinus];
    populationLN[1] = model->tCytotoxicLymphNode[stepKMinus];
    populationLN[2] = model->tHelperLymphNode[stepKMinus];
    populationLN[3] = model->bCellLymphNode[stepKMinus];
    populationLN[4] = model->plasmaCellLymphNode[stepKMinus];
    populationLN[5] = model->antibodyLymphNode[stepKMinus];

    float *solutionLN;
    solutionLN = EquationsLymphNode(*model, populationLN, stepPos);

    // Execute Euler
    verifyValues(*model, model->dendriticLymphNode[stepKMinus], stepPos, "k minus - DC lymph node");
    verifyValues(*model, model->tCytotoxicLymphNode[stepKMinus], stepPos, "k minus - CD8 T lymph node");
    verifyValues(*model, model->tHelperLymphNode[stepKMinus], stepPos, "k minus - CD4 T lymph node");
    verifyValues(*model, model->bCellLymphNode[stepKMinus], stepPos, "k minus - B cell lymph node");
    verifyValues(*model, model->plasmaCellLymphNode[stepKMinus], stepPos, "k minus - Plasma cell lymph node");
    verifyValues(*model, model->antibodyLymphNode[stepKMinus], stepPos, "k minus - Antibody lymph node");

    model->dendriticLymphNode[stepKPlus] = model->dendriticLymphNode[stepKMinus] + model->ht * solutionLN[0];
    model->tCytotoxicLymphNode[stepKPlus] = model->tCytotoxicLymphNode[stepKMinus] + model->ht * solutionLN[1];
    model->tHelperLymphNode[stepKPlus] = model->tHelperLymphNode[stepKMinus] + model->ht * solutionLN[2];
    model->bCellLymphNode[stepKPlus] = model->bCellLymphNode[stepKMinus] + model->ht * solutionLN[3];
    model->plasmaCellLymphNode[stepKPlus] = model->plasmaCellLymphNode[stepKMinus] + model->ht * solutionLN[4];
    model->antibodyLymphNode[stepKPlus] = model->antibodyLymphNode[stepKMinus] + model->ht * solutionLN[5];
    free(solutionLN);

    int intervalPoints = (int)(model->tSize / model->numPointsLN);
    if (stepPos % intervalPoints)
    {
        int posSave = stepPos / intervalPoints;
        model->dendriticLymphNodeSavedPoints[posSave] = model->dendriticLymphNode[stepKPlus];
        model->tCytotoxicLymphNodeSavedPoints[posSave] = model->tCytotoxicLymphNode[stepKPlus];
        model->tHelperLymphNodeSavedPoints[posSave] = model->tHelperLymphNode[stepKPlus];
        model->bCellLymphNodeSavedPoints[posSave] = model->bCellLymphNode[stepKPlus];
        model->plasmaCellLymphNodeSavedPoints[posSave] = model->plasmaCellLymphNode[stepKPlus];
        model->antibodyLymphNodeSavedPoints[posSave] = model->antibodyLymphNode[stepKPlus];
    }
    verifyValues(*model, model->dendriticLymphNode[stepKPlus], stepPos, "DC lymph node");
    verifyValues(*model, model->tCytotoxicLymphNode[stepKPlus], stepPos, "CD8 T lymph node");
    verifyValues(*model, model->tHelperLymphNode[stepKPlus], stepPos, "CD4 T lymph node");
    verifyValues(*model, model->bCellLymphNode[stepKPlus], stepPos, "B cell lymph node");
    verifyValues(*model, model->plasmaCellLymphNode[stepKPlus], stepPos, "Plasma cell lymph node");
    verifyValues(*model, model->antibodyLymphNode[stepKPlus], stepPos, "Antibody lymph node");
}

__device__ __constant__ float upperNeumannBC, lowerNeumannBC, leftNeumannBC, rightNeumannBC, constHx, constHt;
__device__ __constant__ int constXSize;
const int threadsPerBlock = 256;
const int numBlocks = 256;

__global__ void kernelPDE(structParameters *devParams, int kTime, float *tCytoSumVessel, float *activatedDCSumVessel, float *antibodySumVessel, float *devActivatedDCLymphNode, float *devAntibodyLymphNode, float *devTCytotoxicLymphNode, float *devThetaPV, float *devThetaBV, float *devMicrogliaKMinus, float *devMicrogliaKPlus, float *devTCytotoxicKMinus, float *devTCytotoxicKPlus, float *devAntibodyKMinus, float *devAntibodyKPlus, float *devConventionalDCKMinus, float *devConventionalDCKPlus, float *devActivatedDCKMinus, float *devActivatedDCKPlus, float *devOligodendrocyteKMinus, float *devOligodendrocyteKPlus)
{
    int thrIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int vesselIdx = threadIdx.x;
    int line = (int)thrIdx / constXSize;
    int column = thrIdx % constXSize;

    __shared__ float tCytoSumVesselBlock[threadsPerBlock];
    __shared__ float conventionalDCSumVesselBlock[threadsPerBlock];
    __shared__ float antibodySumVesselBlock[threadsPerBlock];
    for (int i = 0; i < threadsPerBlock; i++)
    {
        tCytoSumVesselBlock[i] = 0;
        conventionalDCSumVesselBlock[i] = 0;
        antibodySumVesselBlock[i] = 0;
    }
    while (thrIdx < constXSize * constXSize)
    {
        line = (int)thrIdx / constXSize;
        column = thrIdx % constXSize;

        // Define gradient ODCs
        float valIPlus = (line != constXSize - 1) ? devOligodendrocyteKMinus[thrIdx + constXSize] : devOligodendrocyteKMinus[thrIdx];
        float valJPlus = (column != constXSize - 1) ? devOligodendrocyteKMinus[thrIdx + 1] : devOligodendrocyteKMinus[thrIdx];
        float valIMinus = (line != 0) ? devOligodendrocyteKMinus[thrIdx - constXSize] : devOligodendrocyteKMinus[thrIdx];
        float valJMinus = (column != 0) ? devOligodendrocyteKMinus[thrIdx - 1] : devOligodendrocyteKMinus[thrIdx];

        float gradientOdcI = (float)(valIPlus - valIMinus) / (float)(2 * constHx);
        float gradientOdcJ = (float)(valJPlus - valJMinus) / (float)(2 * constHx);

        // Diffusion and Chemotaxis Mic

        valIPlus = (line != constXSize - 1) ? devMicrogliaKMinus[thrIdx + constXSize] : devMicrogliaKMinus[thrIdx] - (float)(2 * constHx * lowerNeumannBC);
        valJPlus = (column != constXSize - 1) ? devMicrogliaKMinus[thrIdx + 1] : devMicrogliaKMinus[thrIdx] - (float)(2 * constHx * rightNeumannBC);
        valIMinus = (line != 0) ? devMicrogliaKMinus[thrIdx - constXSize] : devMicrogliaKMinus[thrIdx] - (float)(2 * constHx * upperNeumannBC);
        valJMinus = (column != 0) ? devMicrogliaKMinus[thrIdx - 1] : devMicrogliaKMinus[thrIdx] - (float)(2 * constHx * leftNeumannBC);

        float microgliaDiffusion = 0;
        float microgliaChemotaxis = 0;
        CalculateDiffusion(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devMicrogliaKMinus[thrIdx], &microgliaDiffusion);
        CalculateChemottaxis(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devMicrogliaKMinus[thrIdx],
                             devParams->avgMic, gradientOdcI, gradientOdcJ, &microgliaChemotaxis);
        microgliaChemotaxis *= devParams->chi;
        microgliaDiffusion *= devParams->micDiffusion;
        // Diffusion and Chemotaxis CDC

        valIPlus = (line != constXSize - 1) ? devConventionalDCKMinus[thrIdx + constXSize] : devConventionalDCKMinus[thrIdx] - (float)(2 * constHx * lowerNeumannBC);
        valJPlus = (column != constXSize - 1) ? devConventionalDCKMinus[thrIdx + 1] : devConventionalDCKMinus[thrIdx] - (float)(2 * constHx * rightNeumannBC);
        valIMinus = (line != 0) ? devConventionalDCKMinus[thrIdx - constXSize] : devConventionalDCKMinus[thrIdx] - (float)(2 * constHx * upperNeumannBC);
        valJMinus = (column != 0) ? devConventionalDCKMinus[thrIdx - 1] : devConventionalDCKMinus[thrIdx] - (float)(2 * constHx * leftNeumannBC);

        float conventionalDcDiffusion = 0;
        float conventionalDcChemotaxis = 0;
        CalculateDiffusion(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devConventionalDCKMinus[thrIdx], &conventionalDcDiffusion);
        CalculateChemottaxis(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devConventionalDCKMinus[thrIdx],
                             devParams->avgDc, gradientOdcI, gradientOdcJ, &conventionalDcChemotaxis);
        conventionalDcChemotaxis *= devParams->chi;
        conventionalDcDiffusion *= devParams->cDcDiffusion;

        // Difussion and Chemotaxis CD8T

        valIPlus = (line != constXSize - 1) ? devTCytotoxicKMinus[thrIdx + constXSize] : devTCytotoxicKMinus[thrIdx] - (float)(2 * constHx * lowerNeumannBC);
        valJPlus = (column != constXSize - 1) ? devTCytotoxicKMinus[thrIdx + 1] : devTCytotoxicKMinus[thrIdx] - (float)(2 * constHx * rightNeumannBC);
        valIMinus = (line != 0) ? devTCytotoxicKMinus[thrIdx - constXSize] : devTCytotoxicKMinus[thrIdx] - (float)(2 * constHx * upperNeumannBC);
        valJMinus = (column != 0) ? devTCytotoxicKMinus[thrIdx - 1] : devTCytotoxicKMinus[thrIdx] - (float)(2 * constHx * leftNeumannBC);

        float tCytotoxicDiffusion = 0;
        float tCytotoxicChemotaxis = 0;
        CalculateDiffusion(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devTCytotoxicKMinus[thrIdx], &tCytotoxicDiffusion);
        CalculateChemottaxis(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devTCytotoxicKMinus[thrIdx],
                             devParams->avgT, gradientOdcI, gradientOdcJ, &tCytotoxicChemotaxis);
        tCytotoxicChemotaxis *= devParams->chi;
        tCytotoxicDiffusion *= devParams->tCytoDiffusion;

        // Difussion ADC

        valIPlus = (line != constXSize - 1) ? devActivatedDCKMinus[thrIdx + constXSize] : devActivatedDCKMinus[thrIdx] - (float)(2 * constHx * lowerNeumannBC);
        valJPlus = (column != constXSize - 1) ? devActivatedDCKMinus[thrIdx + 1] : devActivatedDCKMinus[thrIdx] - (float)(2 * constHx * rightNeumannBC);
        valIMinus = (line != 0) ? devActivatedDCKMinus[thrIdx - constXSize] : devActivatedDCKMinus[thrIdx] - (float)(2 * constHx * upperNeumannBC);
        valJMinus = (column != 0) ? devActivatedDCKMinus[thrIdx - 1] : devActivatedDCKMinus[thrIdx] - (float)(2 * constHx * leftNeumannBC);

        float activatedDCDiffusion = 0;
        CalculateDiffusion(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devActivatedDCKMinus[thrIdx], &activatedDCDiffusion);
        activatedDCDiffusion *= devParams->aDcDiffusion;

        // Difussion Antibody

        valIPlus = (line != constXSize - 1) ? devAntibodyKMinus[thrIdx + constXSize] : devAntibodyKMinus[thrIdx] - (float)(2 * constHx * lowerNeumannBC);
        valJPlus = (column != constXSize - 1) ? devAntibodyKMinus[thrIdx + 1] : devAntibodyKMinus[thrIdx] - (float)(2 * constHx * rightNeumannBC);
        valIMinus = (line != 0) ? devAntibodyKMinus[thrIdx - constXSize] : devAntibodyKMinus[thrIdx] - (float)(2 * constHx * upperNeumannBC);
        valJMinus = (column != 0) ? devAntibodyKMinus[thrIdx - 1] : devAntibodyKMinus[thrIdx] - (float)(2 * constHx * leftNeumannBC);

        float antibodyDiffusion = 0;
        CalculateDiffusion(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devAntibodyKMinus[thrIdx], &antibodyDiffusion);
        antibodyDiffusion *= devParams->antibodyDiffusion;

        //*******************************************Solving Tissue equations*****************************************************

        // Microglia update
        float microgliaReaction = devParams->muMic * devMicrogliaKMinus[thrIdx] * (devParams->avgMic - devMicrogliaKMinus[thrIdx]);
        float microgliaClearance = devParams->cMic * devMicrogliaKMinus[thrIdx];

        devMicrogliaKPlus[thrIdx] = devMicrogliaKMinus[thrIdx] +
                                    constHt * (microgliaDiffusion - microgliaChemotaxis + microgliaReaction - microgliaClearance);

        // Conventional DC update
        float conventionalDcReaction = devParams->muCDc * devOligodendrocyteKMinus[thrIdx] * (devParams->avgDc - devConventionalDCKMinus[thrIdx]);
        float conventionalDcActivation = devParams->bD * devConventionalDCKMinus[thrIdx] * devOligodendrocyteKMinus[thrIdx];
        float conventionalDcClearance = devParams->cCDc * devConventionalDCKMinus[thrIdx];

        devConventionalDCKPlus[thrIdx] = devConventionalDCKMinus[thrIdx] +
                                         constHt * (conventionalDcDiffusion - conventionalDcChemotaxis - conventionalDcClearance + conventionalDcReaction - conventionalDcActivation);

        // Activated DC update
        float activatedDcClearance = devParams->cADc * devActivatedDCKMinus[thrIdx];
        float activatedDcMigration = devThetaPV[thrIdx] * devParams->gammaD * (*devActivatedDCLymphNode - devActivatedDCKMinus[thrIdx]);

        devActivatedDCKPlus[thrIdx] = devActivatedDCKMinus[thrIdx] + constHt * (activatedDCDiffusion + conventionalDcActivation + activatedDcMigration - activatedDcClearance);

        // CD8 T update
        float tCytotoxicMigration = devThetaBV[thrIdx] * devParams->gammaT * (*devTCytotoxicLymphNode - devTCytotoxicKMinus[thrIdx]);

        devTCytotoxicKPlus[thrIdx] = devTCytotoxicKMinus[thrIdx] + constHt * (tCytotoxicDiffusion - tCytotoxicChemotaxis + tCytotoxicMigration);

        // Antibody update
        float resultFFuncMic = 0;
        fFunc(devMicrogliaKMinus[thrIdx], devParams->avgMic, &resultFFuncMic);
        float odcAntibodyMicrogliaFagocitosis = devParams->lambAntMic * devAntibodyKMinus[thrIdx] * (devParams->avgOdc - devOligodendrocyteKMinus[thrIdx]) * resultFFuncMic;
        float antibodyMigration = devThetaBV[thrIdx] * devParams->gammaAntibody * (*devAntibodyLymphNode - devAntibodyKMinus[thrIdx]);

        devAntibodyKPlus[thrIdx] = devAntibodyKMinus[thrIdx] + constHt * (antibodyDiffusion + antibodyMigration - odcAntibodyMicrogliaFagocitosis);

        // Oligodendrocytes update
        float result = 0, result1 = 0;
        fFunc(devMicrogliaKMinus[thrIdx], devParams->avgMic, &result);
        fFunc(devTCytotoxicKMinus[thrIdx], devParams->avgT, &result1);
        float odcMicrogliaFagocitosis = devParams->rM * result * (devParams->avgOdc - devOligodendrocyteKMinus[thrIdx]);
        float odcTCytotoxicApoptosis = devParams->rT * result1 * (devParams->avgOdc - devOligodendrocyteKMinus[thrIdx]);

        devOligodendrocyteKPlus[thrIdx] = devOligodendrocyteKMinus[thrIdx] + constHt * (odcAntibodyMicrogliaFagocitosis + odcMicrogliaFagocitosis + odcTCytotoxicApoptosis);

        if (devThetaBV[thrIdx] == 1)
        {
            tCytoSumVesselBlock[vesselIdx] += devTCytotoxicKPlus[thrIdx];
            antibodySumVesselBlock[vesselIdx] += devAntibodyKPlus[thrIdx];
        }
        if (devThetaPV[thrIdx] == 1)
        {
            conventionalDCSumVesselBlock[vesselIdx] += devActivatedDCKPlus[thrIdx];
        }
        thrIdx += gridDim.x * blockDim.x;
    }
    __syncthreads();
    int i = blockDim.x / 2;
    while (i != 0)
    {
        if (vesselIdx < i)
        {
            tCytoSumVesselBlock[vesselIdx] += tCytoSumVesselBlock[vesselIdx + i];
            conventionalDCSumVesselBlock[vesselIdx] += conventionalDCSumVesselBlock[vesselIdx + i];
            antibodySumVesselBlock[vesselIdx] += antibodySumVesselBlock[vesselIdx + i];
        }
        __syncthreads();
        i /= 2;
    }
    if (vesselIdx == 0)
    {
        tCytoSumVessel[blockIdx.x] = tCytoSumVesselBlock[0];
        activatedDCSumVessel[blockIdx.x] = conventionalDCSumVesselBlock[0];
        antibodySumVessel[blockIdx.x] = antibodySumVesselBlock[0];
    }
}

void DeleteModel(structModel *model){
    printf("Deleting model..\n");
    for (int index = 0; index < BUFFER; index++)
    {
        free(model->microglia[index]);
        free(model->oligodendrocyte[index]);
        free(model->tCytotoxic[index]);
        free(model->antibody[index]);
        free(model->conventionalDc[index]);
        free(model->activatedDc[index]);
    }
    free(model->microglia);
    free(model->oligodendrocyte);
    free(model->tCytotoxic);
    free(model->antibody);
    free(model->conventionalDc);
    free(model->activatedDc);

    free(model->thetaPV);
    free(model->thetaBV);
    // definir lymph node
    free(model->dendriticLymphNodeSavedPoints);
    free(model->tCytotoxicLymphNodeSavedPoints);
    free(model->tHelperLymphNodeSavedPoints);
    free(model->antibodyLymphNodeSavedPoints);
    free(model->bCellLymphNodeSavedPoints);
    free(model->plasmaCellLymphNodeSavedPoints);

    free(model->dendriticLymphNode);
    free(model->tCytotoxicLymphNode);
    free(model->tHelperLymphNode);
    free(model->antibodyLymphNode);
    free(model->bCellLymphNode);
    free(model->plasmaCellLymphNode);
    printf("Deleting done!\n");
}

void RunModel(structModel *model)
{
    // Save IC
    WriteFiles(*model, model->oligodendrocyte[0], model->microglia[0], model->tCytotoxic[0], model->antibody[0], model->conventionalDc[0], model->activatedDc[0], 0);

    float *activatedDCVessel, *tCytotoxicVessel, *antibodyVessel;

    float *devThetaPV, *devThetaBV, *devActivatedDCVessel, *devTCytotoxicVessel, *devAntibodyVessel, *devActivatedDCLymphNode, *devAntibodyLymphNode, *devTCytotoxicLymphNode, *devMicrogliaKMinus, *devMicrogliaKPlus, *devTCytotoxicKMinus, *devTCytotoxicKPlus, *devAntibodyKMinus, *devAntibodyKPlus, *devConventionalDCKMinus, *devConventionalDCKPlus, *devActivatedDCKMinus, *devActivatedDCKPlus, *devOligodendrocytesDCKMinus, *devOligodendrocytesDCKPlus;

    hipMalloc((void **)&devThetaPV, model->xSize * model->xSize * sizeof(float));
    hipMalloc((void **)&devThetaBV, model->xSize * model->xSize * sizeof(float));

    hipMalloc((void **)&devOligodendrocytesDCKMinus, model->xSize * model->xSize * sizeof(float));
    hipMalloc((void **)&devOligodendrocytesDCKPlus, model->xSize * model->xSize * sizeof(float));

    hipMalloc((void **)&devMicrogliaKMinus, model->xSize * model->xSize * sizeof(float));
    hipMalloc((void **)&devMicrogliaKPlus, model->xSize * model->xSize * sizeof(float));

    hipMalloc((void **)&devTCytotoxicKMinus, model->xSize * model->xSize * sizeof(float));
    hipMalloc((void **)&devTCytotoxicKPlus, model->xSize * model->xSize * sizeof(float));

    hipMalloc((void **)&devAntibodyKMinus, model->xSize * model->xSize * sizeof(float));
    hipMalloc((void **)&devAntibodyKPlus, model->xSize * model->xSize * sizeof(float));

    hipMalloc((void **)&devConventionalDCKMinus, model->xSize * model->xSize * sizeof(float));
    hipMalloc((void **)&devConventionalDCKPlus, model->xSize * model->xSize * sizeof(float));

    hipMalloc((void **)&devActivatedDCKMinus, model->xSize * model->xSize * sizeof(float));
    hipMalloc((void **)&devActivatedDCKPlus, model->xSize * model->xSize * sizeof(float));

    hipMemcpy(devThetaBV, model->thetaBV, model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devThetaPV, model->thetaPV, model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(devOligodendrocytesDCKMinus, model->oligodendrocyte[0], model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devMicrogliaKMinus, model->microglia[0], model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devTCytotoxicKMinus, model->tCytotoxic[0], model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devAntibodyKMinus, model->antibody[0], model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devConventionalDCKMinus, model->conventionalDc[0], model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devActivatedDCKMinus, model->activatedDc[0], model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);

    structParameters *devParams;

    // se der errado passar parametro por parametro (tentar com memoria de constantes)
    hipMalloc((void **)&devParams, sizeof(structParameters));
    hipMemcpy(devParams, &model->parametersModel, sizeof(structParameters), hipMemcpyHostToDevice);

    hipMalloc((void **)&devActivatedDCLymphNode, sizeof(float));
    hipMalloc((void **)&devAntibodyLymphNode, sizeof(float));
    hipMalloc((void **)&devTCytotoxicLymphNode, sizeof(float));

    hipMalloc((void **)&devActivatedDCVessel, numBlocks * sizeof(float));
    hipMalloc((void **)&devAntibodyVessel, numBlocks * sizeof(float));
    hipMalloc((void **)&devTCytotoxicVessel, numBlocks * sizeof(float));
    // Inicializar os constant com os valores
    int stepKMinus = 0, stepKPlus;

    float auxAdcPV = 0.0, auxAntibodyBV = 0.0, auxTCytotoxicBV = 0.0;

    float bc = 0.0;

    hipMemcpyToSymbol(HIP_SYMBOL(upperNeumannBC), &bc, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(lowerNeumannBC), &bc, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(leftNeumannBC), &bc, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(rightNeumannBC), &bc, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(constHt), &model->ht, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(constHx), &model->hx, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(constXSize), &model->xSize, sizeof(float));

    int devKTime;
    hipMalloc((void **)&devKTime, sizeof(int));

    for (int kTime = 1; kTime <= model->tSize; kTime++)
    {
        auxAdcPV = 0.0, auxAntibodyBV = 0.0, auxTCytotoxicBV = 0.0;
        // solve lymphnode
        SolverLymphNode(model, kTime);
        // printf("Passou do linfonodo no tempo %d", kTime);
        stepKPlus = kTime % 2;
        // copiar LN pra GPU
        hipMemcpy(devActivatedDCLymphNode, &model->dendriticLymphNode[stepKPlus], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(devAntibodyLymphNode, &model->antibodyLymphNode[stepKPlus], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(devTCytotoxicLymphNode, &model->tCytotoxicLymphNode[stepKPlus], sizeof(float), hipMemcpyHostToDevice);

        hipMemcpy(&devKTime, &kTime, sizeof(int), hipMemcpyHostToDevice);

        if (stepKPlus % 2 == 1)
            kernelPDE<<<numBlocks, threadsPerBlock>>>(devParams, devKTime, devTCytotoxicVessel, devActivatedDCVessel, devAntibodyVessel, devActivatedDCLymphNode, devAntibodyLymphNode, devTCytotoxicLymphNode, devThetaPV, devThetaBV, devMicrogliaKMinus, devMicrogliaKPlus, devTCytotoxicKMinus, devTCytotoxicKPlus, devAntibodyKMinus, devAntibodyKPlus, devConventionalDCKMinus, devConventionalDCKPlus, devActivatedDCKMinus, devActivatedDCKPlus, devOligodendrocytesDCKMinus, devOligodendrocytesDCKPlus);
        else
            kernelPDE<<<numBlocks, threadsPerBlock>>>(devParams, devKTime, devTCytotoxicVessel, devActivatedDCVessel, devAntibodyVessel, devActivatedDCLymphNode, devAntibodyLymphNode, devTCytotoxicLymphNode, devThetaPV, devThetaBV, devMicrogliaKPlus, devMicrogliaKMinus, devTCytotoxicKPlus, devTCytotoxicKMinus, devAntibodyKPlus, devAntibodyKMinus, devConventionalDCKPlus, devConventionalDCKMinus, devActivatedDCKPlus, devActivatedDCKMinus, devOligodendrocytesDCKPlus, devOligodendrocytesDCKMinus);

        if (kTime % model->intervalFigures == 0 || kTime == model->tSize)
        {
            if (stepKPlus % 2 == 1)
            {
                hipMemcpy(model->oligodendrocyte[stepKPlus], devOligodendrocytesDCKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->microglia[stepKPlus], devMicrogliaKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->tCytotoxic[stepKPlus], devTCytotoxicKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->antibody[stepKPlus], devAntibodyKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->conventionalDc[stepKPlus], devConventionalDCKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->activatedDc[stepKPlus], devActivatedDCKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
            }
            else
            {
                hipMemcpy(model->oligodendrocyte[stepKPlus], devOligodendrocytesDCKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->microglia[stepKPlus], devMicrogliaKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->tCytotoxic[stepKPlus], devTCytotoxicKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->antibody[stepKPlus], devAntibodyKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->conventionalDc[stepKPlus], devConventionalDCKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->activatedDc[stepKPlus], devActivatedDCKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
            }

            WriteFiles(*model, model->oligodendrocyte[stepKPlus], model->microglia[stepKPlus], model->tCytotoxic[stepKPlus], model->antibody[stepKPlus], model->conventionalDc[stepKPlus], model->activatedDc[stepKPlus], kTime);
        }
        // Copia do device para o host as integrais do tecido
        activatedDCVessel = (float *)calloc(numBlocks, sizeof(float));
        antibodyVessel = (float *)calloc(numBlocks, sizeof(float));
        tCytotoxicVessel = (float *)calloc(numBlocks, sizeof(float));

        hipMemcpy(activatedDCVessel, devActivatedDCVessel, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(antibodyVessel, devAntibodyVessel, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(tCytotoxicVessel, devTCytotoxicVessel, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
        for (int pos = 0; pos < numBlocks; pos++)
        {
            auxAdcPV += activatedDCVessel[pos];
            auxAntibodyBV += antibodyVessel[pos];
            auxTCytotoxicBV += tCytotoxicVessel[pos];
        }
        model->tCytotoxicTissueVessels = auxTCytotoxicBV * model->hx * model->hx / model->parametersModel.V_BV;
        model->antibodyTissueVessels = auxAntibodyBV * model->hx * model->hx / model->parametersModel.V_BV;
        model->activatedDCTissueVessels = auxAdcPV * model->hx * model->hx / model->parametersModel.V_PV;

        free(activatedDCVessel);
        free(antibodyVessel);
        free(tCytotoxicVessel);
        stepKMinus += 1;
        stepKMinus = stepKMinus % 2;
    }
    printf("Computation Done!!\n");
    printf("Saving results...\n\n");
    WriteLymphNodeFiles(*model, model->dendriticLymphNodeSavedPoints, model->tHelperLymphNodeSavedPoints, model->tCytotoxicLymphNodeSavedPoints, model->bCellLymphNodeSavedPoints, model->plasmaCellLymphNodeSavedPoints, model->antibodyLymphNodeSavedPoints);
    PlotResults(*model);
    printf("Deleting cuda memory...\n");

    hipFree(devThetaPV);
    hipFree(devThetaBV);

    hipFree(devOligodendrocytesDCKMinus);
    hipFree(devOligodendrocytesDCKPlus);

    hipFree(devMicrogliaKMinus);
    hipFree(devMicrogliaKPlus);

    hipFree(devTCytotoxicKMinus);
    hipFree(devTCytotoxicKPlus);

    hipFree(devAntibodyKMinus);
    hipFree(devAntibodyKPlus);

    hipFree(devConventionalDCKMinus);
    hipFree(devConventionalDCKPlus);

    hipFree(devActivatedDCKMinus);
    hipFree(devActivatedDCKPlus);

    hipFree(devParams);

    hipFree(devActivatedDCLymphNode);
    hipFree(devAntibodyLymphNode);
    hipFree(devTCytotoxicLymphNode);

    hipFree(devActivatedDCVessel);
    hipFree(devAntibodyVessel);
    hipFree(devTCytotoxicVessel);

    hipFree(&devKTime);

    printf("CUDA memory deleted!\n");
    DeleteModel(model);
}