#include "hip/hip_runtime.h"
#include "model.h"
#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

void InitialConditionTissueMicroglia(structModel* model){
    for(int k = 0; k < model->xSize*model->xSize; k++){
        int i = (int)k/model->xSize;
        int j = k%model->xSize;
        if(pow((i-(int)(model->xSize/2)),2) + pow((j-(int)(model->xSize/2)),2) < 5 / (model->hx * model->hx)){
            model->microglia[0][k] = (float)model->parametersModel.avgMic/3;
        }
    }
}

void InitialConditionLymphNode(structModel *model, float dendriticLN, float thelperLN, float tcytotoxicLN, float bcellLN, float plasmacellLN, float antibodyLN)
{
    model->dendriticLymphNode[0] = dendriticLN;
    model->tHelperLymphNode[0] = thelperLN;
    model->tCytotoxicLymphNode[0] = tcytotoxicLN;
    model->bCellLymphNode[0] = bcellLN;
    model->plasmaCellLymphNode[0] = plasmacellLN;
    model->antibodyLymphNode[0] = antibodyLN;
}

int VerifyCFL(structParameters parametersModel, float ht, float hx){
    if(parametersModel.micDiffusion*ht/(hx*hx) < 0.25 && parametersModel.cDcDiffusion*ht/(hx*hx) < 0.25 && parametersModel.aDcDiffusion*ht/(hx*hx) < 0.25 && parametersModel.tCytoDiffusion*ht/(hx*hx) < 0.25 && parametersModel.chi*ht/hx < 0.5)
        return 1;
    return 0;
}

void WritePopulation(structModel model, float *population, char *fileName, char *bufferTime)
{
    FILE *file;
    file = fopen(fileName, "w");
    if(file != NULL){
        int k = 0;
        while (k < model.xSize*model.xSize){
            int i = k;
            while (i < k + model.xSize){
                fprintf(file, "%f ", population[i]);
                i++;
            }
            fprintf(file,"\n");
            k+=model.xSize;
        }
        fclose(file);
    }else{
        printf("Error lymph node file\n");
        exit(0);
    }
}

void WritePopulationLymphNode(structModel model, float *population, char *fileName)
{
    FILE *file;
    file = fopen(fileName, "w");
    if(file != NULL){
        for(int i=0;i<model.numPointsLN;i++){
            fprintf(file, "%f\n", population[i]);
        }
        fclose(file);
    }else{
        printf("Error matrix file\n");
        exit(0);
    }
}

void WriteLymphNodeFiles(structModel model, float *dendritic, float *tHelper, float *tCytotoxic, float *bCell, float *plasmaCell, float *antibody)
{
    WritePopulationLymphNode(model, dendritic, "./result/dendritic.txt");
    WritePopulationLymphNode(model, tHelper, "./result/tHelper.txt");
    WritePopulationLymphNode(model, tCytotoxic, "./result/tCyto.txt");
    WritePopulationLymphNode(model, bCell, "./result/bCell.txt");
    WritePopulationLymphNode(model, plasmaCell, "./result/plasmaCell.txt");
    WritePopulationLymphNode(model, antibody, "./result/antibody.txt");

    char buffer[10];
    char command[40] = {};
    strcat(command, "python3 plotLymphNode.py ");
    snprintf(buffer, sizeof(buffer), "%d", model.tFinal);
    strcat(command, buffer);
    strcat(command, " ");
    snprintf(buffer, sizeof(buffer), "%f", (model.tSize / model.numPointsLN) * model.ht);
    strcat(command, buffer);
    system(command);
}

void WriteFiles(structModel model, float *oligodendrocyte, float *microglia, float *tCytotoxic, float *antibody, float *conventionalDC, float *activatedDC, float time)
{
    char buffer[10];
    float day = time * model.ht;

    snprintf(buffer, sizeof(buffer), "%.1f", day);

    char pathOligodendrocytes[200] = "./result/matrix/oligo";
    strcat(pathOligodendrocytes, buffer);
    strcat(pathOligodendrocytes, ".txt");
    WritePopulation(model, oligodendrocyte, pathOligodendrocytes, buffer);

    char pathMicroglia[200] = "./result/matrix/microglia";
    strcat(pathMicroglia, buffer);
    strcat(pathMicroglia, ".txt");
    WritePopulation(model, microglia, pathMicroglia, buffer);

    char pathTCyto[200] = "./result/matrix/tCyto";
    strcat(pathTCyto, buffer);
    strcat(pathTCyto, ".txt");
    WritePopulation(model, tCytotoxic, pathTCyto, buffer);

    char pathAntibody[200] = "./result/matrix/antibody";
    strcat(pathAntibody, buffer);
    strcat(pathAntibody, ".txt");
    WritePopulation(model, antibody, pathAntibody, buffer);

    char pathConventionalDC[200] = "./result/matrix/conventionalDC";
    strcat(pathConventionalDC, buffer);
    strcat(pathConventionalDC, ".txt");
    WritePopulation(model, conventionalDC, pathConventionalDC, buffer);

    char pathActivatedDC[200] = "./result/matrix/activatedDC";
    strcat(pathActivatedDC, buffer);
    strcat(pathActivatedDC, ".txt");
    WritePopulation(model, activatedDC, pathActivatedDC, buffer);
}

void PlotResults(structModel model)
{
    char buffer[10];
    char command[70] = {};
    strcat(command, "python3 plotMatrices.py ");
    snprintf(buffer, sizeof(buffer), "%d", model.xFinal);
    strcat(command, buffer);
    strcat(command, " ");
    snprintf(buffer, sizeof(buffer), "%f", model.hx);
    strcat(command, buffer);
    strcat(command, " ");
    snprintf(buffer, sizeof(buffer), "%d", model.tFinal);
    strcat(command, buffer);
    strcat(command, " ");
    snprintf(buffer, sizeof(buffer), "%d", model.tSize/model.intervalFigures);
    strcat(command, buffer);
    system(command);
}

__device__ void PreventionOverCrowdingTerm(float populationPoint, float avgValue, float *result)
{
    *result = populationPoint / (populationPoint + avgValue);
}

__device__ void UpDownWind(float frontPoint, float rearPoint, float avgValue, float *result)
{
    float resultF;
    PreventionOverCrowdingTerm(frontPoint, avgValue, result);
    PreventionOverCrowdingTerm(rearPoint, avgValue, &resultF);
    *result = *result - resultF;
}

__device__ void CalculateChemottaxis(float hx, float frontJPoint, float rearJPoint, float frontIPoint, float rearIPoint, float ijPoint,
                                     float avgValue, float gradientOdcI, float gradientOdcJ, float *result)
{
    float gradientPopulationI, gradientPopulationJ;
    if (gradientOdcI < 0)
    {
        UpDownWind(frontIPoint, ijPoint, avgValue, &gradientPopulationI);
        gradientPopulationI = gradientPopulationI / (float)hx;
    }
    else
    {
        UpDownWind(ijPoint, rearIPoint, avgValue, &gradientPopulationI);
        gradientPopulationI = gradientPopulationI / (float)hx;
    }
    if (gradientOdcJ < 0)
    {
        UpDownWind(frontJPoint, ijPoint, avgValue, &gradientPopulationJ);
        gradientPopulationJ = gradientPopulationJ / (float)hx;
    }
    else
    {
        UpDownWind(ijPoint, rearJPoint, avgValue, &gradientPopulationJ);
        gradientPopulationJ = gradientPopulationJ / (float)hx;
    }

    *result = gradientOdcI * gradientPopulationI + gradientOdcJ * gradientPopulationJ;
}

__device__ void CalculateDiffusion(float hx, float frontJPoint, float rearJPoint, float frontIPoint, float rearIPoint, float ijPoint, float *result)
{
    *result = (float)(frontIPoint + frontJPoint - 4 * ijPoint + rearIPoint + rearJPoint) / (float)(hx * hx);
}

__device__ void fFunc(float valuePopulation, float avgPopulation, float *result)
{
    *result = valuePopulation * valuePopulation / (float)(valuePopulation + avgPopulation);
}

void WriteBVPV(structModel *model, float *thetaBV, float *thetaPV){
    FILE *fileBV;
    fileBV = fopen("./result/bv.txt", "w");
    FILE *filePV;
    filePV = fopen("./result/pv.txt", "w");
    int k = 0;
    if(fileBV != NULL && filePV != NULL){
        while (k < model->xSize*model->xSize){
            int i = k;
            while (i < k + model->xSize){
                fprintf(fileBV, "%f ", thetaBV[i]);
                fprintf(filePV, "%f ", thetaPV[i]);
                i++;
            }
            fprintf(fileBV,"\n");
            fprintf(filePV,"\n");
            k+=model->xSize;
        }
        fclose(fileBV);
        fclose(filePV);
    }else{
        printf("Error matrix file\n");
        exit(0);
    }
    char buffer[10];
    char command[200] = {};
    strcat(command, "python3 plotBVPV.py ");
    snprintf(buffer, sizeof(buffer), "%d", model->xFinal);
    strcat(command, buffer);
    strcat(command, " ");
    snprintf(buffer, sizeof(buffer), "%f", model->hx);
    strcat(command, buffer);
    // system(command);
}

void DefineBVPV(structModel *model){
    int randomVal;
    for(int k = 0; k < model->xSize*model->xSize; k++){
        int i = (int)k/model->xSize;
        int j = k%model->xSize;
        randomVal = rand() % 100;
        if(randomVal <10){
            model->parametersModel.V_BV++;
            model->parametersModel.V_PV++;
            model->thetaBV[k] = 1;
            if(j != model->xSize-1)
                model->thetaPV[k+1] = 1;
            else
                model->thetaPV[k-model->xSize+1] = 1;
        }
    }
    model->parametersModel.V_BV = model->parametersModel.V_BV * model->hx * model->hx;
    model->parametersModel.V_PV = model->parametersModel.V_PV * model->hx * model->hx;
    printf("bv = %f, pv = %f \n", model->parametersModel.V_BV, model->parametersModel.V_PV);
    WriteBVPV(model, model->thetaBV, model->thetaPV);
}

structModel ModelInitialize(structParameters params, float ht, float hx, float time, float space, int numFigs, int numPointsLN, int numStepsLN, int saveFigs){
    structModel model;
    srand(2);
    model.parametersModel = params;
    if(!VerifyCFL(model.parametersModel, ht, hx)){
        printf("Falhou CFL!!\n");
        exit(1);
    }
    model.parametersModel = params;
    model.numFigs = numFigs;
    model.numPointsLN = numPointsLN;
    model.numStepsLN = numStepsLN;

    model.ht = ht;
    model.hx = hx;
    model.tFinal = time;
    model.xFinal = space;
    model.tSize = (int)(time/ht);
    model.xSize = (int)(space/hx);
    model.intervalFigures = (int)model.tSize/numFigs;
    model.saveFigs = saveFigs;

    model.microglia = (float**)malloc(BUFFER * sizeof(float*));
    model.oligodendrocyte = (float**)malloc(BUFFER * sizeof(float*));
    model.tCytotoxic = (float**)malloc(BUFFER * sizeof(float*));
    model.antibody = (float**)malloc(BUFFER * sizeof(float*));
    model.conventionalDc = (float**)malloc(BUFFER * sizeof(float*));
    model.activatedDc = (float**)malloc(BUFFER * sizeof(float*));

    model.activatedDCTissueVessels = 0;
    model.tCytotoxicTissueVessels = 0;
    model.antibodyTissueVessels = 0;

    for (int index=0;index<BUFFER;++index){
        model.microglia[index] = (float*)calloc(model.xSize*model.xSize, sizeof(float));
        model.oligodendrocyte[index] = (float*)calloc(model.xSize*model.xSize, sizeof(float));
        model.tCytotoxic[index] = (float*)calloc(model.xSize*model.xSize, sizeof(float));
        model.conventionalDc[index] = (float*)calloc(model.xSize*model.xSize, sizeof(float));
        model.activatedDc[index] = (float*)calloc(model.xSize*model.xSize, sizeof(float));
        model.antibody[index] = (float*)calloc(model.xSize*model.xSize, sizeof(float));
    }
    //definir BV e PV
    model.thetaPV = (float*)calloc(model.xSize*model.xSize, sizeof(float));
    model.thetaBV = (float*)calloc(model.xSize*model.xSize, sizeof(float));
    DefineBVPV(&model);
    //definir lymph node
    model.dendriticLymphNodeSavedPoints = (float*)calloc(model.numPointsLN, sizeof(float));
    model.tCytotoxicLymphNodeSavedPoints = (float*)calloc(model.numPointsLN, sizeof(float));
    model.tHelperLymphNodeSavedPoints = (float*)calloc(model.numPointsLN, sizeof(float));
    model.antibodyLymphNodeSavedPoints = (float*)calloc(model.numPointsLN, sizeof(float));
    model.bCellLymphNodeSavedPoints = (float*)calloc(model.numPointsLN, sizeof(float));
    model.plasmaCellLymphNodeSavedPoints = (float*)calloc(model.numPointsLN, sizeof(float));

    model.dendriticLymphNode = (float*)calloc(2, sizeof(float));
    model.tCytotoxicLymphNode = (float*)calloc(2, sizeof(float));
    model.tHelperLymphNode = (float*)calloc(2, sizeof(float));
    model.antibodyLymphNode = (float*)calloc(2, sizeof(float));
    model.bCellLymphNode = (float*)calloc(2, sizeof(float));
    model.plasmaCellLymphNode = (float*)calloc(2, sizeof(float));    

    float dendriticLN = 0.0, thelperLN = params.stableTHelper, tcytotoxicLN = params.stableTCytotoxic, bcellLN = params.stableB, plasmacellLN = 0.0, antibodyLN = 0.0;
    InitialConditionLymphNode(&model, dendriticLN, thelperLN, tcytotoxicLN, bcellLN, plasmacellLN, antibodyLN);
    InitialConditionTissueMicroglia(&model);
    return model;
}

void verifyValues(structModel model, float value, int time, char* populationName){
    if(value < 0 ||  isnanf(value)){
        printf("Error: %s = (%f) :: time = %f\n", populationName, value, time*model.ht);
        exit(0);
    }
}

void verifyDerivate(structModel model, float value, int time, char* populationName){
    if(isnanf(value)){
        printf("Error: %s = (%f) :: time = %f\n", populationName, value, time*model.ht);
        exit(0);
    }
}

float* EquationsLymphNode(structModel model, float* populationLN, int stepPos){
    float* result =(float *)malloc(sizeof(float)*6);
    
    float dcLN = populationLN[0];
    float tCytoLN = populationLN[1];
    float tHelperLN = populationLN[2];
    float bCellLN = populationLN[3];
    float plasmaCellLN = populationLN[4];
    float antibodyLN = populationLN[5];

    //Describe equations

    //Dendritic cell
    float activatedDcMigration = model.parametersModel.gammaD * (model.activatedDCTissueVessels - dcLN) * (float)(model.parametersModel.V_PV/model.parametersModel.V_LN);
    float activatedDcClearance = model.parametersModel.cDl * dcLN;
    result[0] = activatedDcMigration - activatedDcClearance;

    //T Cytotoxic
    float tCytoActivation = model.parametersModel.bTCytotoxic * (model.parametersModel.rhoTCytotoxic*tCytoLN*dcLN - tCytoLN*dcLN);
    float tCytoHomeostasis = model.parametersModel.alphaTCytotoxic * (model.parametersModel.stableTCytotoxic - tCytoLN);
    float tCytoMigration = model.parametersModel.gammaT * (tCytoLN - model.tCytotoxicTissueVessels) * (float)(model.parametersModel.V_BV/model.parametersModel.V_LN) - (1 * model.parametersModel.epslon_x);
    result[1] = tCytoActivation + tCytoHomeostasis - tCytoMigration;

    //T Helper
    float tHelperActivation = model.parametersModel.bTHelper * (model.parametersModel.rhoTHelper * tHelperLN * dcLN - tHelperLN * dcLN);
    float tHelperHomeostasis = model.parametersModel.alphaTHelper * (model.parametersModel.stableTHelper - tHelperLN);
    float tHelperDispendure = model.parametersModel.bRho * dcLN * tHelperLN * bCellLN;
    result[2] = tHelperActivation + tHelperHomeostasis - tHelperDispendure;

    //B Cell
    float bCellActivation = model.parametersModel.bRhoB * (model.parametersModel.rhoB * tHelperLN * dcLN - tHelperLN * dcLN * bCellLN);
    float bcellHomeostasis = model.parametersModel.alphaB * (model.parametersModel.stableB - bCellLN);
    result[3] = bcellHomeostasis + bCellActivation;

    //Plasma Cells
    float plasmaActivation = model.parametersModel.bRhoP * (model.parametersModel.rhoP * tHelperLN * dcLN * bCellLN);
    float plasmaHomeostasis = model.parametersModel.alphaP * (model.parametersModel.stableP - plasmaCellLN);
    result[4] = plasmaHomeostasis + plasmaActivation;

    //Antibody
    float antibodyProduction = model.parametersModel.rhoAntibody * plasmaCellLN;
    float antibodyDecayment = model.parametersModel.cF * antibodyLN;
    float antibodyMigration = model.parametersModel.gammaAntibody * (antibodyLN - model.antibodyTissueVessels) * (float)(model.parametersModel.V_BV/model.parametersModel.V_LN);
    result[5] = antibodyProduction - antibodyMigration - antibodyDecayment;

    return result;
}

void SolverLymphNode(structModel *model, int stepPos){
    float populationLN[6];
    int stepKPlus = (stepPos%(2*model->numStepsLN))/model->numStepsLN;
    int stepKMinus = !(stepKPlus && 1);
    populationLN[0] = model->dendriticLymphNode[stepKMinus];
    populationLN[1] = model->tCytotoxicLymphNode[stepKMinus];
    populationLN[2] = model->tHelperLymphNode[stepKMinus];
    populationLN[3] = model->bCellLymphNode[stepKMinus];
    populationLN[4] = model->plasmaCellLymphNode[stepKMinus];
    populationLN[5] = model->antibodyLymphNode[stepKMinus];
    
    float* solutionLN;
    solutionLN = EquationsLymphNode(*model, populationLN, stepPos);
    
    float htLN = model->ht*model->numStepsLN;

    //Execute Euler 
    model->dendriticLymphNode[stepKPlus] = model->dendriticLymphNode[stepKMinus] + htLN*solutionLN[0];
    model->tCytotoxicLymphNode[stepKPlus] = model->tCytotoxicLymphNode[stepKMinus] + htLN*solutionLN[1];
    model->tHelperLymphNode[stepKPlus] = model->tHelperLymphNode[stepKMinus] + htLN*solutionLN[2];
    model->bCellLymphNode[stepKPlus] = model->bCellLymphNode[stepKMinus] + htLN*solutionLN[3];
    model->plasmaCellLymphNode[stepKPlus] = model->plasmaCellLymphNode[stepKMinus] + htLN*solutionLN[4];
    model->antibodyLymphNode[stepKPlus] = model->antibodyLymphNode[stepKMinus] + htLN*solutionLN[5];
    free(solutionLN);

    int intervalPoints = (int)(model->tSize/model->numPointsLN);
    if(stepPos%intervalPoints){
        int posSave = stepPos/intervalPoints;
        model->dendriticLymphNodeSavedPoints[posSave] = model->dendriticLymphNode[stepKPlus];
        model->tCytotoxicLymphNodeSavedPoints[posSave] = model->tCytotoxicLymphNode[stepKPlus];
        model->tHelperLymphNodeSavedPoints[posSave] = model->tHelperLymphNode[stepKPlus];
        model->bCellLymphNodeSavedPoints[posSave] = model->bCellLymphNode[stepKPlus];
        model->plasmaCellLymphNodeSavedPoints[posSave] = model->plasmaCellLymphNode[stepKPlus];
        model->antibodyLymphNodeSavedPoints[posSave] = model->antibodyLymphNode[stepKPlus];
    }
}

void SavingData(structModel model){
    float totalMic = 0, totalODC = 0, totalCDC = 0, totalADC = 0, totalIGG = 0, totalCD8 = 0;
    for(int kPos = 0; kPos < model.xSize*model.xSize; kPos++){
        totalMic += model.microglia[0][kPos];
        totalODC += model.oligodendrocyte[0][kPos];
        totalCDC += model.conventionalDc[0][kPos];
        totalADC += model.activatedDc[0][kPos];
        totalCD8 += model.tCytotoxic[0][kPos];
        totalIGG += model.antibody[0][kPos];
    }
    FILE *file;
    file = fopen("dataExecution.txt", "w");
    if(file != NULL){
        fprintf(file, "Execution Time of Kernel = %f secs\n", model.execTimeKernel);
        fprintf(file, "Execution Time of Lymph Node = %f secs\n", model.elapsedTimeLymphNode);
        fprintf(file, "Execution Time of Copies Device to Host = %f secs\n", model.elapsedTimeCopiesDeviceToHost);
        fprintf(file, "Execution Time of Copies Host to Device = %f secs\n", model.elapsedTimeCopiesHostToDevice);
        fprintf(file, "Days = %d - Space = %d - ht = %f, hx = %f, Ht_JumpStep = %d\n", model.tFinal, model.xFinal, model.ht, model.hx, model.numStepsLN);
        fprintf(file, "Lymph node populations\n");
        fprintf(file, "DC = %f, TCD8 = %f, TCD4 = %f, B Cell = %f, Plasma cell = %f, IgG = %f\n", model.dendriticLymphNodeSavedPoints[model.numPointsLN-1], model.tCytotoxicLymphNodeSavedPoints[model.numPointsLN-1], model.tHelperLymphNodeSavedPoints[model.numPointsLN-1], model.bCellLymphNodeSavedPoints[model.numPointsLN-1], model.plasmaCellLymphNodeSavedPoints[model.numPointsLN-1], model.antibodyLymphNodeSavedPoints[model.numPointsLN-1]);
        fprintf(file, "Tissue populations\n");
        fprintf(file, "ODC = %f, Microglia = %f, ConventionalDC = %f, ActivatedDC = %f, TCD8 = %f, IgG = %f\n", totalODC, totalMic, totalCDC, totalADC, totalCD8, totalIGG);    
        fprintf(file, "Parameters\n");
        fprintf(file, "micDiffusion  = %f, antibodyDiffusion = %f, cDcDiffusion = %f, aDcDiffusion = %f, tCytoDiffusion = %f, chi = %f, muCDc = %f, muMic = %f, \
        rM = %f, rT = %f, lambAntMic = %f, bD = %f, gammaD = %f, gammaAntibody = %f, gammaT = %f,  avgT = %f, avgDc = %f, avgMic = %f, avgOdc = %f,  cMic = %f, \
        cCDc = %f, cADc = %f, cDl = %f, cF = %f, alphaTHelper = %f, alphaTCytotoxic = %f, alphaB = %f, alphaP = %f, bTHelper = %f, bTCytotoxic = %f, bRho = %f, \
        bRhoB = %f, bRhoP = %f, rhoTHelper = %f, rhoTCytotoxic = %f, rhoB = %f, rhoP = %f, rhoAntibody = %f, stableTHelper = %f, stableTCytotoxic = %f, \
        stableB = %f, stableP = %f, V_LN = %d, V_BV = %f, V_PV = %f\n",
        model.parametersModel.micDiffusion, model.parametersModel.antibodyDiffusion, model.parametersModel.cDcDiffusion, model.parametersModel.aDcDiffusion, \
        model.parametersModel.tCytoDiffusion, model.parametersModel.chi, model.parametersModel.muCDc, model.parametersModel.muMic, model.parametersModel.rM, \
        model.parametersModel.rT, model.parametersModel.lambAntMic, model.parametersModel.bD, model.parametersModel.gammaD, model.parametersModel.gammaAntibody, \
        model.parametersModel.gammaT,  model.parametersModel.avgT, model.parametersModel.avgDc, model.parametersModel.avgMic, model.parametersModel.avgOdc, \
        model.parametersModel.cMic, model.parametersModel.cCDc, model.parametersModel.cADc, model.parametersModel.cDl, model.parametersModel.cF, \
        model.parametersModel.alphaTHelper, model.parametersModel.alphaTCytotoxic, model.parametersModel.alphaB, model.parametersModel.alphaP, \
        model.parametersModel.bTHelper, model.parametersModel.bTCytotoxic, model.parametersModel.bRho, model.parametersModel.bRhoB, model.parametersModel.bRhoP,\
        model.parametersModel.rhoTHelper, model.parametersModel.rhoTCytotoxic, model.parametersModel.rhoB, model.parametersModel.rhoP,\
        model.parametersModel.rhoAntibody, model.parametersModel.stableTHelper, model.parametersModel.stableTCytotoxic, model.parametersModel.stableB,\
        model.parametersModel.stableP, model.parametersModel.V_LN, model.parametersModel.V_BV, model.parametersModel.V_PV);
        fclose(file);
    }
}

__device__ __constant__ float upperNeumannBC, lowerNeumannBC, leftNeumannBC, rightNeumannBC, constHx, constHt, consthx2;
__device__ __constant__ int constXSize;
__device__ __constant__ structParameters modelParams;
const int threadsPerBlock = 64;
const int numBlocks = 16;

__global__ void kernelPDE(int kTime, float *tCytoSumVessel, float *activatedDCSumVessel, float *antibodySumVessel, float *devActivatedDCLymphNode, float *devAntibodyLymphNode, float *devTCytotoxicLymphNode, float *devThetaPV, float *devThetaBV, float *devMicrogliaKMinus, float *devMicrogliaKPlus, float *devTCytotoxicKMinus, float *devTCytotoxicKPlus, float *devAntibodyKMinus, float *devAntibodyKPlus, float *devConventionalDCKMinus, float *devConventionalDCKPlus, float *devActivatedDCKMinus, float *devActivatedDCKPlus, float *devOligodendrocyteKMinus, float *devOligodendrocyteKPlus)
{
    int thrIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int vesselIdx = threadIdx.x;
    int line = (int)thrIdx / constXSize;
    int column = thrIdx % constXSize;
    float devOligodendrocyteKMinusThrIdx, devMicrogliaKMinusThrIdx, devConventionalDCKMinusThrIdx, devTCytotoxicKMinusThrIdx, devActivatedDCKMinusThrIdx, devAntibodyKMinusThrIdx;
    float avgOdcMinusODC, diffusionODC;
    __shared__ float tCytoSumVesselBlock[threadsPerBlock];
    __shared__ float activatedDCSumVesselBlock[threadsPerBlock];
    __shared__ float antibodySumVesselBlock[threadsPerBlock];
    for (int i = 0; i < threadsPerBlock; i++)
    {
        tCytoSumVesselBlock[i] = 0;
        activatedDCSumVesselBlock[i] = 0;
        antibodySumVesselBlock[i] = 0;
    }
    while (thrIdx < constXSize * constXSize)
    {
        devOligodendrocyteKMinusThrIdx = devOligodendrocyteKMinus[thrIdx];
        devMicrogliaKMinusThrIdx = devMicrogliaKMinus[thrIdx];
        devConventionalDCKMinusThrIdx = devConventionalDCKMinus[thrIdx];
        devTCytotoxicKMinusThrIdx = devTCytotoxicKMinus[thrIdx];
        devActivatedDCKMinusThrIdx = devActivatedDCKMinus[thrIdx];
        devAntibodyKMinusThrIdx = devAntibodyKMinus[thrIdx];
        line = (int)thrIdx / constXSize;
        column = thrIdx % constXSize;

        // Define gradient ODCs
        float valIPlus = (line != constXSize - 1) ? devOligodendrocyteKMinus[thrIdx + constXSize] : devOligodendrocyteKMinus[thrIdx - constXSize];
        float valJPlus = (column != constXSize - 1) ? devOligodendrocyteKMinus[thrIdx + 1] : devOligodendrocyteKMinus[thrIdx - 1];
        float valIMinus = (line != 0) ? devOligodendrocyteKMinus[thrIdx - constXSize] : devOligodendrocyteKMinus[thrIdx + constXSize];
        float valJMinus = (column != 0) ? devOligodendrocyteKMinus[thrIdx - 1] : devOligodendrocyteKMinus[thrIdx + 1];

        float gradientOdcI = (float)(valIPlus - valIMinus) / (float)(constHx*2);
        float gradientOdcJ = (float)(valJPlus - valJMinus) / (float)(constHx*2);

        CalculateDiffusion(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devOligodendrocyteKMinusThrIdx, &diffusionODC);
        // Diffusion and Chemotaxis Mic

        valIPlus = (line != constXSize - 1) ? devMicrogliaKMinus[thrIdx + constXSize] : devMicrogliaKMinus[thrIdx] + ( devMicrogliaKMinus[thrIdx] / (devMicrogliaKMinus[thrIdx] + modelParams.avgMic) ) * constHx * gradientOdcI/modelParams.micDiffusion;
        valJPlus = (column != constXSize - 1) ? devMicrogliaKMinus[thrIdx + 1] : devMicrogliaKMinus[thrIdx] + ( devMicrogliaKMinus[thrIdx] / (devMicrogliaKMinus[thrIdx] + modelParams.avgMic) ) * constHx * gradientOdcJ/modelParams.micDiffusion;
        valIMinus = (line != 0) ? devMicrogliaKMinus[thrIdx - constXSize] : devMicrogliaKMinus[thrIdx] - ( devMicrogliaKMinus[thrIdx] / (devMicrogliaKMinus[thrIdx] + modelParams.avgMic) ) * constHx * gradientOdcI/modelParams.micDiffusion;
        valJMinus = (column != 0) ? devMicrogliaKMinus[thrIdx - 1] : devMicrogliaKMinus[thrIdx] - ( devMicrogliaKMinus[thrIdx] / (devMicrogliaKMinus[thrIdx] + modelParams.avgMic) ) * constHx * gradientOdcJ/modelParams.micDiffusion;

        float microgliaDiffusion = 0;
        float microgliaChemotaxis = 0;
        CalculateDiffusion(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devMicrogliaKMinusThrIdx, &microgliaDiffusion);
        CalculateChemottaxis(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devMicrogliaKMinusThrIdx,
                             modelParams.avgMic, gradientOdcI, gradientOdcJ, &microgliaChemotaxis);
        microgliaChemotaxis += diffusionODC * devMicrogliaKMinusThrIdx / (devMicrogliaKMinusThrIdx + modelParams.avgMic);
        microgliaChemotaxis *= modelParams.chi;
        microgliaDiffusion *= modelParams.micDiffusion;
        // Diffusion and Chemotaxis CDC

        valIPlus = (line != constXSize - 1) ? devConventionalDCKMinus[thrIdx + constXSize] : devConventionalDCKMinus[thrIdx] + ( devConventionalDCKMinus[thrIdx] / (devConventionalDCKMinus[thrIdx] + modelParams.avgDc) ) * constHx * gradientOdcI/modelParams.cDcDiffusion;
        valJPlus = (column != constXSize - 1) ? devConventionalDCKMinus[thrIdx + 1] : devConventionalDCKMinus[thrIdx] + ( devConventionalDCKMinus[thrIdx] / (devConventionalDCKMinus[thrIdx] + modelParams.avgDc) ) * constHx * gradientOdcJ/modelParams.cDcDiffusion;
        valIMinus = (line != 0) ? devConventionalDCKMinus[thrIdx - constXSize] : devConventionalDCKMinus[thrIdx] - ( devConventionalDCKMinus[thrIdx] / (devConventionalDCKMinus[thrIdx] + modelParams.avgDc) ) * constHx * gradientOdcI/modelParams.cDcDiffusion;
        valJMinus = (column != 0) ? devConventionalDCKMinus[thrIdx - 1] : devConventionalDCKMinus[thrIdx] - ( devConventionalDCKMinus[thrIdx] / (devConventionalDCKMinus[thrIdx] + modelParams.avgDc) ) * constHx * gradientOdcJ/modelParams.cDcDiffusion;

        float conventionalDcDiffusion = 0;
        float conventionalDcChemotaxis = 0;
        CalculateDiffusion(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devConventionalDCKMinusThrIdx, &conventionalDcDiffusion);
        CalculateChemottaxis(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devConventionalDCKMinusThrIdx,
                             modelParams.avgDc, gradientOdcI, gradientOdcJ, &conventionalDcChemotaxis);
        conventionalDcChemotaxis += diffusionODC * devConventionalDCKMinusThrIdx / (devConventionalDCKMinusThrIdx + modelParams.avgDc);
        conventionalDcChemotaxis *= modelParams.chi;
        conventionalDcDiffusion *= modelParams.cDcDiffusion;

        // Difussion and Chemotaxis CD8T

        valIPlus = (line != constXSize - 1) ? devTCytotoxicKMinus[thrIdx + constXSize] : devTCytotoxicKMinus[thrIdx] + ( devTCytotoxicKMinus[thrIdx] / (devTCytotoxicKMinus[thrIdx] + modelParams.avgT) ) * constHx * gradientOdcI/modelParams.tCytoDiffusion;
        valJPlus = (column != constXSize - 1) ? devTCytotoxicKMinus[thrIdx + 1] : devTCytotoxicKMinus[thrIdx] + ( devTCytotoxicKMinus[thrIdx] / (devTCytotoxicKMinus[thrIdx] + modelParams.avgT) ) * constHx * gradientOdcJ/modelParams.tCytoDiffusion;
        valIMinus = (line != 0) ? devTCytotoxicKMinus[thrIdx - constXSize] : devTCytotoxicKMinus[thrIdx] - ( devTCytotoxicKMinus[thrIdx] / (devTCytotoxicKMinus[thrIdx] + modelParams.avgT) ) * constHx * gradientOdcI/modelParams.tCytoDiffusion;
        valJMinus = (column != 0) ? devTCytotoxicKMinus[thrIdx - 1] : devTCytotoxicKMinus[thrIdx] - ( devTCytotoxicKMinus[thrIdx] / (devTCytotoxicKMinus[thrIdx] + modelParams.avgT) ) * constHx * gradientOdcJ/modelParams.tCytoDiffusion;

        float tCytotoxicDiffusion = 0;
        float tCytotoxicChemotaxis = 0;
        CalculateDiffusion(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devTCytotoxicKMinusThrIdx, &tCytotoxicDiffusion);
        CalculateChemottaxis(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devTCytotoxicKMinusThrIdx,
                             modelParams.avgT, gradientOdcI, gradientOdcJ, &tCytotoxicChemotaxis);
        tCytotoxicChemotaxis += diffusionODC * devTCytotoxicKMinusThrIdx / (devTCytotoxicKMinusThrIdx + modelParams.avgT);
        tCytotoxicChemotaxis *= modelParams.chi;
        tCytotoxicDiffusion *= modelParams.tCytoDiffusion;

        // Difussion ADC

        valIPlus = (line != constXSize - 1) ? devActivatedDCKMinus[thrIdx + constXSize] : devActivatedDCKMinus[thrIdx - constXSize] - (float)(constHx*2 * lowerNeumannBC);
        valJPlus = (column != constXSize - 1) ? devActivatedDCKMinus[thrIdx + 1] : devActivatedDCKMinus[thrIdx - 1] - (float)(constHx*2 * rightNeumannBC);
        valIMinus = (line != 0) ? devActivatedDCKMinus[thrIdx - constXSize] : devActivatedDCKMinus[thrIdx + constXSize] - (float)(constHx*2 * upperNeumannBC);
        valJMinus = (column != 0) ? devActivatedDCKMinus[thrIdx - 1] : devActivatedDCKMinus[thrIdx + 1] - (float)(constHx*2 * leftNeumannBC);

        float activatedDCDiffusion = 0;
        CalculateDiffusion(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devActivatedDCKMinusThrIdx, &activatedDCDiffusion);
        activatedDCDiffusion *= modelParams.aDcDiffusion;

        // Difussion Antibody

        valIPlus = (line != constXSize - 1) ? devAntibodyKMinus[thrIdx + constXSize] : devAntibodyKMinus[thrIdx - constXSize] - (float)(constHx*2 * lowerNeumannBC);
        valJPlus = (column != constXSize - 1) ? devAntibodyKMinus[thrIdx + 1] : devAntibodyKMinus[thrIdx - 1] - (float)(constHx*2 * rightNeumannBC);
        valIMinus = (line != 0) ? devAntibodyKMinus[thrIdx - constXSize] : devAntibodyKMinus[thrIdx + constXSize] - (float)(constHx*2 * upperNeumannBC);
        valJMinus = (column != 0) ? devAntibodyKMinus[thrIdx - 1] : devAntibodyKMinus[thrIdx + 1] - (float)(constHx*2 * leftNeumannBC);

        float antibodyDiffusion = 0;
        CalculateDiffusion(constHx, valJPlus, valJMinus, valIPlus, valIMinus, devAntibodyKMinusThrIdx, &antibodyDiffusion);
        antibodyDiffusion *= modelParams.antibodyDiffusion;

        //*******************************************Solving Tissue equations*****************************************************
        
        // Microglia update
        float microgliaReaction = modelParams.muMic * devMicrogliaKMinusThrIdx * (modelParams.avgMic - devMicrogliaKMinusThrIdx);
        float microgliaClearance = modelParams.cMic * devMicrogliaKMinusThrIdx;

        devMicrogliaKPlus[thrIdx] = devMicrogliaKMinusThrIdx +
                                    constHt * (microgliaDiffusion - microgliaChemotaxis + microgliaReaction - microgliaClearance);

        // Conventional DC update
        float conventionalDcReaction = modelParams.muCDc * devOligodendrocyteKMinusThrIdx * (modelParams.avgDc - devConventionalDCKMinusThrIdx);
        float conventionalDcActivation = modelParams.bD * devConventionalDCKMinusThrIdx * devOligodendrocyteKMinusThrIdx;
        float conventionalDcClearance = modelParams.cCDc * devConventionalDCKMinusThrIdx;

        devConventionalDCKPlus[thrIdx] = devConventionalDCKMinusThrIdx +
                                         constHt * (conventionalDcDiffusion - conventionalDcChemotaxis - conventionalDcClearance + conventionalDcReaction - conventionalDcActivation);

        // Activated DC update
        float activatedDcClearance = modelParams.cADc * devActivatedDCKMinusThrIdx;
        float activatedDcMigration = devThetaPV[thrIdx] * modelParams.gammaD * (*devActivatedDCLymphNode - devActivatedDCKMinusThrIdx);

        devActivatedDCKPlus[thrIdx] = devActivatedDCKMinusThrIdx + constHt * (activatedDCDiffusion + conventionalDcActivation + activatedDcMigration - activatedDcClearance);

        // CD8 T update
        float tCytotoxicMigration = devThetaBV[thrIdx] * modelParams.gammaT * (*devTCytotoxicLymphNode - devTCytotoxicKMinusThrIdx);

        devTCytotoxicKPlus[thrIdx] = devTCytotoxicKMinusThrIdx + constHt * (tCytotoxicDiffusion - tCytotoxicChemotaxis + tCytotoxicMigration);

        // Antibody update
        float resultFFuncMic = 0;
        fFunc(devMicrogliaKMinusThrIdx, modelParams.avgMic, &resultFFuncMic);
        avgOdcMinusODC = modelParams.avgOdc - devOligodendrocyteKMinusThrIdx;
        float odcAntibodyMicrogliaFagocitosis = modelParams.lambAntMic * devAntibodyKMinusThrIdx * avgOdcMinusODC * resultFFuncMic;
        float antibodyMigration = devThetaBV[thrIdx] * modelParams.gammaAntibody * (*devAntibodyLymphNode - devAntibodyKMinusThrIdx);

        devAntibodyKPlus[thrIdx] = devAntibodyKMinusThrIdx + constHt * (antibodyDiffusion + antibodyMigration - odcAntibodyMicrogliaFagocitosis);

        // Oligodendrocytes update
        float result = 0;
        fFunc(devTCytotoxicKMinusThrIdx, modelParams.avgT, &result);
        float odcMicrogliaFagocitosis = modelParams.rM * resultFFuncMic * avgOdcMinusODC;
        float odcTCytotoxicApoptosis = modelParams.rT * result * avgOdcMinusODC;

        devOligodendrocyteKPlus[thrIdx] = devOligodendrocyteKMinusThrIdx + constHt * (odcAntibodyMicrogliaFagocitosis + odcMicrogliaFagocitosis + odcTCytotoxicApoptosis);

        if (devThetaBV[thrIdx] == 1)
        {
            tCytoSumVesselBlock[vesselIdx] += devTCytotoxicKPlus[thrIdx];
            antibodySumVesselBlock[vesselIdx] += devAntibodyKPlus[thrIdx];
        }
        if (devThetaPV[thrIdx] == 1)
        {
            activatedDCSumVesselBlock[vesselIdx] += devActivatedDCKPlus[thrIdx];
        }
        thrIdx += gridDim.x * blockDim.x;
    }
    __syncthreads();
    int i = blockDim.x / 2;
    while (i != 0)
    {
        if (vesselIdx < i)
        {
            tCytoSumVesselBlock[vesselIdx] += tCytoSumVesselBlock[vesselIdx + i];
            activatedDCSumVesselBlock[vesselIdx] += activatedDCSumVesselBlock[vesselIdx + i];
            antibodySumVesselBlock[vesselIdx] += antibodySumVesselBlock[vesselIdx + i];
        }
        __syncthreads();
        i /= 2;
    }
    if (vesselIdx == 0)
    {
        tCytoSumVessel[blockIdx.x] = tCytoSumVesselBlock[0];
        activatedDCSumVessel[blockIdx.x] = activatedDCSumVesselBlock[0];
        antibodySumVessel[blockIdx.x] = antibodySumVesselBlock[0];
    }
}

void RunModel(structModel *model)
{
    // Save IC
    if(model->saveFigs)
        WriteFiles(*model, model->oligodendrocyte[0], model->microglia[0], model->tCytotoxic[0], model->antibody[0], model->conventionalDc[0], model->activatedDc[0], 0);

    clock_t start, end;
    float elapsedTimeLymphNode = 0, elapsedTimeCopiesDeviceToHost = 0, elapsedTimeCopiesHostToDevice = 0;
    float elapsedTimeKernel = 0, elapsedTimeKernelAux = 0;
    hipEvent_t startKernel, stopKernel;
    hipEventCreate(&startKernel);
    hipEventCreate(&stopKernel);

    float *activatedDCVessel, *tCytotoxicVessel, *antibodyVessel;

    float *devThetaPV, *devThetaBV, *devActivatedDCVessel, *devTCytotoxicVessel, *devAntibodyVessel, *devActivatedDCLymphNode, *devAntibodyLymphNode, *devTCytotoxicLymphNode, *devMicrogliaKMinus, *devMicrogliaKPlus, *devTCytotoxicKMinus, *devTCytotoxicKPlus, *devAntibodyKMinus, *devAntibodyKPlus, *devConventionalDCKMinus, *devConventionalDCKPlus, *devActivatedDCKMinus, *devActivatedDCKPlus, *devOligodendrocytesDCKMinus, *devOligodendrocytesDCKPlus;

    hipMalloc((void **)&devThetaPV, model->xSize * model->xSize * sizeof(float));
    hipMalloc((void **)&devThetaBV, model->xSize * model->xSize * sizeof(float));

    hipMalloc((void **)&devOligodendrocytesDCKMinus, model->xSize * model->xSize * sizeof(float));
    hipMalloc((void **)&devOligodendrocytesDCKPlus, model->xSize * model->xSize * sizeof(float));

    hipMalloc((void **)&devMicrogliaKMinus, model->xSize * model->xSize * sizeof(float));
    hipMalloc((void **)&devMicrogliaKPlus, model->xSize * model->xSize * sizeof(float));

    hipMalloc((void **)&devTCytotoxicKMinus, model->xSize * model->xSize * sizeof(float));
    hipMalloc((void **)&devTCytotoxicKPlus, model->xSize * model->xSize * sizeof(float));

    hipMalloc((void **)&devAntibodyKMinus, model->xSize * model->xSize * sizeof(float));
    hipMalloc((void **)&devAntibodyKPlus, model->xSize * model->xSize * sizeof(float));

    hipMalloc((void **)&devConventionalDCKMinus, model->xSize * model->xSize * sizeof(float));
    hipMalloc((void **)&devConventionalDCKPlus, model->xSize * model->xSize * sizeof(float));

    hipMalloc((void **)&devActivatedDCKMinus, model->xSize * model->xSize * sizeof(float));
    hipMalloc((void **)&devActivatedDCKPlus, model->xSize * model->xSize * sizeof(float));

    hipMemcpy(devThetaBV, model->thetaBV, model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devThetaPV, model->thetaPV, model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(devOligodendrocytesDCKMinus, model->oligodendrocyte[0], model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devMicrogliaKMinus, model->microglia[0], model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devTCytotoxicKMinus, model->tCytotoxic[0], model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devAntibodyKMinus, model->antibody[0], model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devConventionalDCKMinus, model->conventionalDc[0], model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devActivatedDCKMinus, model->activatedDc[0], model->xSize * model->xSize * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&devActivatedDCLymphNode, sizeof(float));
    hipMalloc((void **)&devAntibodyLymphNode, sizeof(float));
    hipMalloc((void **)&devTCytotoxicLymphNode, sizeof(float));

    hipMalloc((void **)&devActivatedDCVessel, numBlocks * sizeof(float));
    hipMalloc((void **)&devAntibodyVessel, numBlocks * sizeof(float));
    hipMalloc((void **)&devTCytotoxicVessel, numBlocks * sizeof(float));
    // Inicializar os constant com os valores
    int stepKMinus = 0, stepKPlus;

    float auxAdcPV = 0.0, auxAntibodyBV = 0.0, auxTCytotoxicBV = 0.0;

    float bc = 0.0;
    float hx2 = model->hx * 2;

    hipMemcpyToSymbol(HIP_SYMBOL(upperNeumannBC), &bc, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(lowerNeumannBC), &bc, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(leftNeumannBC), &bc, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(rightNeumannBC), &bc, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(constHt), &model->ht, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(constHx), &model->hx, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(consthx2), &hx2, sizeof(float));
    
    hipMemcpyToSymbol(HIP_SYMBOL(constXSize), &model->xSize, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(modelParams), &model->parametersModel, sizeof(structParameters));

    int devKTime;
    hipMalloc((void **)&devKTime, sizeof(int));
    activatedDCVessel = (float *)calloc(numBlocks, sizeof(float));
    antibodyVessel = (float *)calloc(numBlocks, sizeof(float));
    tCytotoxicVessel = (float *)calloc(numBlocks, sizeof(float));
    for (int kTime = 1; kTime <= model->tSize; kTime++)
    {	
        // solve lymphnode
        if(kTime%model->numStepsLN == 0){
            // Copia do device para o host as integrais do tecido
            start = clock(); 
            hipMemcpy(activatedDCVessel, devActivatedDCVessel, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(antibodyVessel, devAntibodyVessel, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(tCytotoxicVessel, devTCytotoxicVessel, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
            end = clock();

            elapsedTimeCopiesDeviceToHost += ((float) (end - start)) / CLOCKS_PER_SEC;

            auxAdcPV = 0.0, auxAntibodyBV = 0.0, auxTCytotoxicBV = 0.0;
            for (int pos = 0; pos < numBlocks; pos++)
            {
                auxAdcPV += activatedDCVessel[pos];
                auxAntibodyBV += antibodyVessel[pos];
                auxTCytotoxicBV += tCytotoxicVessel[pos];
            }
            model->tCytotoxicTissueVessels = auxTCytotoxicBV * model->hx * model->hx / model->parametersModel.V_BV;
            model->antibodyTissueVessels = auxAntibodyBV * model->hx * model->hx / model->parametersModel.V_BV;
            model->activatedDCTissueVessels = auxAdcPV * model->hx * model->hx / model->parametersModel.V_PV;  

            start = clock(); 
            SolverLymphNode(model, kTime); 
            end = clock();

            elapsedTimeLymphNode += ((float) (end - start)) / CLOCKS_PER_SEC;
        }        
        
        stepKPlus = kTime % 2;
        // copiar LN pra GPU
        if(kTime%model->numStepsLN == 0){
            start = clock(); 
            hipMemcpy(devActivatedDCLymphNode, &model->dendriticLymphNode[stepKPlus], sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(devAntibodyLymphNode, &model->antibodyLymphNode[stepKPlus], sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(devTCytotoxicLymphNode, &model->tCytotoxicLymphNode[stepKPlus], sizeof(float), hipMemcpyHostToDevice);
            end = clock();
            elapsedTimeCopiesHostToDevice += ((float) (end - start)) / CLOCKS_PER_SEC;
        }
        hipMemcpy(&devKTime, &kTime, sizeof(int), hipMemcpyHostToDevice);

        hipEventRecord(startKernel, 0);
        if (stepKPlus % 2 == 1)
            kernelPDE<<<numBlocks, threadsPerBlock>>>(devKTime, devTCytotoxicVessel, devActivatedDCVessel, devAntibodyVessel, devActivatedDCLymphNode, devAntibodyLymphNode, devTCytotoxicLymphNode, devThetaPV, devThetaBV, devMicrogliaKMinus, devMicrogliaKPlus, devTCytotoxicKMinus, devTCytotoxicKPlus, devAntibodyKMinus, devAntibodyKPlus, devConventionalDCKMinus, devConventionalDCKPlus, devActivatedDCKMinus, devActivatedDCKPlus, devOligodendrocytesDCKMinus, devOligodendrocytesDCKPlus);
        else
            kernelPDE<<<numBlocks, threadsPerBlock>>>(devKTime, devTCytotoxicVessel, devActivatedDCVessel, devAntibodyVessel, devActivatedDCLymphNode, devAntibodyLymphNode, devTCytotoxicLymphNode, devThetaPV, devThetaBV, devMicrogliaKPlus, devMicrogliaKMinus, devTCytotoxicKPlus, devTCytotoxicKMinus, devAntibodyKPlus, devAntibodyKMinus, devConventionalDCKPlus, devConventionalDCKMinus, devActivatedDCKPlus, devActivatedDCKMinus, devOligodendrocytesDCKPlus, devOligodendrocytesDCKMinus);
        hipEventRecord(stopKernel, 0);
        hipEventSynchronize(stopKernel);
        hipEventElapsedTime(&elapsedTimeKernelAux, startKernel, stopKernel);
        elapsedTimeKernel += elapsedTimeKernelAux;
        if (model->saveFigs && kTime % model->intervalFigures == 0)
        {
            if (stepKPlus % 2 == 1)
            {
                hipMemcpy(model->oligodendrocyte[stepKPlus], devOligodendrocytesDCKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->microglia[stepKPlus], devMicrogliaKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->tCytotoxic[stepKPlus], devTCytotoxicKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->antibody[stepKPlus], devAntibodyKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->conventionalDc[stepKPlus], devConventionalDCKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->activatedDc[stepKPlus], devActivatedDCKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
            }
            else
            {
                hipMemcpy(model->oligodendrocyte[stepKPlus], devOligodendrocytesDCKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->microglia[stepKPlus], devMicrogliaKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->tCytotoxic[stepKPlus], devTCytotoxicKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->antibody[stepKPlus], devAntibodyKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->conventionalDc[stepKPlus], devConventionalDCKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(model->activatedDc[stepKPlus], devActivatedDCKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
            }
            WriteFiles(*model, model->oligodendrocyte[stepKPlus], model->microglia[stepKPlus], model->tCytotoxic[stepKPlus], model->antibody[stepKPlus], model->conventionalDc[stepKPlus], model->activatedDc[stepKPlus], kTime);
        }else{
            if (kTime == model->tSize){
                if (stepKPlus % 2 == 1)
                {
                    hipMemcpy(model->oligodendrocyte[stepKPlus], devOligodendrocytesDCKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                    hipMemcpy(model->microglia[stepKPlus], devMicrogliaKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                    hipMemcpy(model->tCytotoxic[stepKPlus], devTCytotoxicKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                    hipMemcpy(model->antibody[stepKPlus], devAntibodyKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                    hipMemcpy(model->conventionalDc[stepKPlus], devConventionalDCKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                    hipMemcpy(model->activatedDc[stepKPlus], devActivatedDCKPlus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                }
                else
                {
                    hipMemcpy(model->oligodendrocyte[stepKPlus], devOligodendrocytesDCKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                    hipMemcpy(model->microglia[stepKPlus], devMicrogliaKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                    hipMemcpy(model->tCytotoxic[stepKPlus], devTCytotoxicKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                    hipMemcpy(model->antibody[stepKPlus], devAntibodyKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                    hipMemcpy(model->conventionalDc[stepKPlus], devConventionalDCKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                    hipMemcpy(model->activatedDc[stepKPlus], devActivatedDCKMinus, model->xSize * model->xSize * sizeof(float), hipMemcpyDeviceToHost);
                }
                if(model->saveFigs)
                    WriteFiles(*model, model->oligodendrocyte[stepKPlus], model->microglia[stepKPlus], model->tCytotoxic[stepKPlus], model->antibody[stepKPlus], model->conventionalDc[stepKPlus], model->activatedDc[stepKPlus], kTime);
            }
        }
        stepKMinus += 1;
        stepKMinus = stepKMinus % 2;
    }
    model->elapsedTimeLymphNode = elapsedTimeLymphNode;
    model->elapsedTimeCopiesDeviceToHost = elapsedTimeCopiesDeviceToHost;
    model->elapsedTimeCopiesHostToDevice = elapsedTimeCopiesHostToDevice;
    model->execTimeKernel = elapsedTimeKernel/1000;
    printf("Computation Done!!\n");
    SavingData(*model);
    if(model->saveFigs){
        printf("Saving results...\n\n");
        WriteLymphNodeFiles(*model, model->dendriticLymphNodeSavedPoints, model->tHelperLymphNodeSavedPoints, model->tCytotoxicLymphNodeSavedPoints, model->bCellLymphNodeSavedPoints, model->plasmaCellLymphNodeSavedPoints, model->antibodyLymphNodeSavedPoints);
        PlotResults(*model);
    }
}