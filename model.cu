#include "hip/hip_runtime.h"
#include "model.h"
#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

void InitialConditionTissueMicroglia(structModel* model){
    for(int k = 0; k < model->xSize*model->xSize; k++){
        int i = (int)k/model->xSize;
        int j = k%model->xSize;
        if(pow((i-(int)(model->xSize/2)),2) + pow((j-(int)(model->xSize/2)),2) < 5){
            model->microglia[0][k] = (float)model->parametersModel.avgMic/3;
        }
    }
}

void InitialConditionLymphNode(structModel* model, float dendriticLN, float thelperLN, float tcytotoxicLN, float bcellLN, float plasmacellLN, float antibodyLN){
    model->dendriticLymphNode[0] = dendriticLN;
    model->tHelperLymphNode[0] = thelperLN;
    model->tCytotoxicLymphNode[0] = tcytotoxicLN;
    model->bCellLymphNode[0] = bcellLN;
    model->plasmaCellLymphNode[0] = plasmacellLN;
    model->antibodyLymphNode[0] = antibodyLN;
}

int VerifyCFL(structParameters parametersModel, float ht, float hx){

    return 0;
}

void WritePopulation(structModel model, float *population, char* fileName, char* bufferTime){
    FILE *file;
    file = fopen(fileName, "w");
    int k = 0;
    while (k < model.xSize*model.xSize){
        int i = k;
        while (i < k + model.xSize){
            fprintf(file, "%f ", population[i]);
            i++;
        }
        fprintf(file,"\n");
        k+=model.xSize;
    }
    fclose(file);
}

void WritePopulationLymphNode(structModel model, float *population, char* fileName){
    FILE *file;
    file = fopen(fileName, "w");
    for(int i=0;i<model.numPointsLN;i++){
        fprintf(file, "%f\n", population[i]);
    }
    fclose(file);
}

void WriteLymphNodeFiles(structModel model, float *dendritic, float *tHelper, float *tCytotoxic, float *bCell, float *plasmaCell, float *antibody){
    WritePopulationLymphNode(model, dendritic, "./result/dendritic.txt");
    WritePopulationLymphNode(model, tHelper, "./result/tHelper.txt");
    WritePopulationLymphNode(model, tCytotoxic, "./result/tCyto.txt");
    WritePopulationLymphNode(model, bCell, "./result/bCell.txt");
    WritePopulationLymphNode(model, plasmaCell, "./result/plasmaCell.txt");
    WritePopulationLymphNode(model, antibody, "./result/antibody.txt");

    char buffer[10];
    char command[40] = {};
    strcat(command, "python3 plotLymphNode.py ");
    snprintf(buffer, sizeof(buffer), "%d", model.tFinal);
    strcat(command, buffer);
    strcat(command, " ");
    snprintf(buffer, sizeof(buffer), "%f", (model.tSize/model.numPointsLN)*model.ht);
    strcat(command, buffer);
    system(command);
}

void WriteFiles(structModel model, float *oligodendrocyte, float *microglia, float *tCytotoxic, float *antibody, float *conventionalDC, float  *activatedDC, float time){
    char buffer[10];
    float day = time * model.ht;
    
    snprintf(buffer, sizeof(buffer), "%.1f", day);
    
    char pathOligodendrocytes[50] = "./result/matrix/oligo";
    strcat(pathOligodendrocytes, buffer);
    strcat(pathOligodendrocytes, ".txt");
    WritePopulation(model, oligodendrocyte, pathOligodendrocytes, buffer);

    char pathMicroglia[50] = "./result/matrix/microglia";
    strcat(pathMicroglia, buffer);
    strcat(pathMicroglia, ".txt");
    WritePopulation(model, microglia, pathMicroglia, buffer);

    char pathTCyto[50] = "./result/matrix/tCyto";
    strcat(pathTCyto, buffer);
    strcat(pathTCyto, ".txt");
    WritePopulation(model, tCytotoxic, pathTCyto, buffer);

    char pathAntibody[50] = "./result/matrix/antibody";
    strcat(pathAntibody, buffer);
    strcat(pathAntibody, ".txt");
    WritePopulation(model, antibody, pathAntibody, buffer);

    char pathConventionalDC[50] = "./result/matrix/conventionalDC";
    strcat(pathConventionalDC, buffer);
    strcat(pathConventionalDC, ".txt");
    WritePopulation(model, conventionalDC, pathConventionalDC, buffer);

    char pathActivatedDC[50] = "./result/matrix/activatedDC";
    strcat(pathActivatedDC, buffer);
    strcat(pathActivatedDC, ".txt");
    WritePopulation(model, activatedDC, pathActivatedDC, buffer);
}   

void PlotResults(structModel model){
    char buffer[10];
    char command[70] = {};
    strcat(command, "python3 plotMatrices.py ");
    snprintf(buffer, sizeof(buffer), "%d", model.xFinal);
    strcat(command, buffer);
    strcat(command, " ");
    snprintf(buffer, sizeof(buffer), "%f", model.hx);
    strcat(command, buffer);
    strcat(command, " ");
    snprintf(buffer, sizeof(buffer), "%d", model.tFinal);
    strcat(command, buffer);
    strcat(command, " ");
    snprintf(buffer, sizeof(buffer), "%d", model.intervalFigures);
    strcat(command, buffer);
    system(command);
}

float AdvectionTerm(float populationPoint, float avgValue){
    return populationPoint/(populationPoint + avgValue);
}

float UpDownWind(float frontPoint, float rearPoint, float avgValue){
    return AdvectionTerm(frontPoint, avgValue) - AdvectionTerm(rearPoint, avgValue);
}

float CalculateChemottaxis(float hx, float frontJPoint, float rearJPoint, float frontIPoint, float rearIPoint, float ijPoint,\
 float avgValue, float gradientOdcI, float gradientOdcJ){
    float gradientPopulationI, gradientPopulationJ;
    if(gradientOdcI<0)
        gradientPopulationI = UpDownWind(frontIPoint, ijPoint, avgValue)/(float)hx;
    else
        gradientPopulationI = UpDownWind(ijPoint, rearIPoint, avgValue)/(float)hx;
    if(gradientOdcJ<0)
        gradientPopulationJ = UpDownWind(frontJPoint, ijPoint, avgValue)/(float)hx;
    else
        gradientPopulationJ = UpDownWind(ijPoint, rearJPoint, avgValue)/(float)hx;

    return gradientOdcI*gradientPopulationI + gradientOdcJ*gradientPopulationJ;
}

float CalculateDiffusion(float hx, float frontJPoint, float rearJPoint, float frontIPoint, float rearIPoint, float ijPoint){
    return (float)(frontIPoint + frontJPoint - 4*ijPoint + rearIPoint + rearJPoint)/(float)(hx*hx);
}

float fFunc(float valuePopulation, float avgPopulation){
    return valuePopulation*valuePopulation/(float)(valuePopulation + avgPopulation);
}

void WriteBVPV(structModel *model, float *thetaBV, float *thetaPV){
    FILE *fileBV;
    fileBV = fopen("./result/bv.txt", "w");
    FILE *filePV;
    filePV = fopen("./result/pv.txt", "w");
    for(int k = 0; k < model->xSize*model->xSize; k++){
        int i = (int)k/model->xSize;
        int j = k%model->xSize;
        fprintf(fileBV, "%f ", thetaBV[k]);
        fprintf(filePV, "%f ", thetaPV[k]);    
        if(k%model->xSize == 0 && k != 0){
            fprintf(fileBV,"\n");
            fprintf(filePV,"\n");
        }
    }
    fclose(fileBV);
    fclose(filePV);
    char buffer[10];
    char command[70] = {};
    strcat(command, "python3 plotBVPV.py ");
    snprintf(buffer, sizeof(buffer), "%d", model->xFinal);
    strcat(command, buffer);
    strcat(command, " ");
    snprintf(buffer, sizeof(buffer), "%f", model->hx);
    strcat(command, buffer);
    // system(command);
}

void DefineBVPV(structModel *model){
    int randomVal;
    for(int k = 0; k < model->xSize*model->xSize; k++){
        int i = (int)k/model->xSize;
        int j = k%model->xSize;
        randomVal = rand() % 100;
        if(randomVal <10){
            model->parametersModel.V_BV++;
            model->parametersModel.V_PV++;
            model->thetaBV[k] = 1;
            if(j != model->xSize-1)
                model->thetaPV[k+1] = 1;
            else
                model->thetaPV[k-model->xSize+1] = 1;
        }
    }
    printf("bv = %d, pv = %d \n", model->parametersModel.V_BV, model->parametersModel.V_PV);
    WriteBVPV(model, model->thetaBV, model->thetaPV);
}



structModel ModelInitialize(structParameters params, float ht, float hx, float time, float space, int numFigs, int numPointsLN){
    structModel model;
    srand(2);

    //Pegar os valores pelos parametros
    model.parametersModel = params;
    model.numFigs = numFigs;
    model.numPointsLN = numPointsLN;
    model.ht = ht;
    model.hx = hx;
    model.tFinal = time;
    model.xFinal = space;
    model.tSize = (int)(time/ht);
    model.xSize = (int)(space/hx);
    model.intervalFigures = (int)model.tSize/numFigs;

    //inicializar dinamicamente todos os vetores do tecido
    model.microglia = (float**)malloc(BUFFER * sizeof(float*));
    model.oligodendrocyte = (float**)malloc(BUFFER * sizeof(float*));
    model.tCytotoxic = (float**)malloc(BUFFER * sizeof(float*));
    model.antibody = (float**)malloc(BUFFER * sizeof(float*));
    model.conventionalDc = (float**)malloc(BUFFER * sizeof(float*));
    model.activatedDc = (float**)malloc(BUFFER * sizeof(float*));
    for (int index=0;index<BUFFER;++index){
        model.microglia[index] = (float*)malloc(model.xSize*model.xSize * sizeof(float));
        model.oligodendrocyte[index] = (float*)malloc(model.xSize*model.xSize * sizeof(float));
        model.tCytotoxic[index] = (float*)malloc(model.xSize*model.xSize * sizeof(float));
        model.antibody[index] = (float*)malloc(model.xSize*model.xSize * sizeof(float));
        model.conventionalDc[index] = (float*)malloc(model.xSize*model.xSize * sizeof(float));
        model.activatedDc[index] = (float*)malloc(model.xSize*model.xSize * sizeof(float));
    }
    //definir BV e PV
    model.thetaPV = (float*)malloc(model.xSize*model.xSize * sizeof(float));
    model.thetaBV = (float*)malloc(model.xSize*model.xSize * sizeof(float));
    DefineBVPV(&model);
    //definir lymph node
    model.dendriticLymphNodeSavedPoints = (float*)malloc(model.numPointsLN * sizeof(float));
    model.tCytotoxicLymphNodeSavedPoints = (float*)malloc(model.numPointsLN * sizeof(float));
    model.tHelperLymphNodeSavedPoints = (float*)malloc(model.numPointsLN * sizeof(float));
    model.antibodyLymphNodeSavedPoints = (float*)malloc(model.numPointsLN * sizeof(float));
    model.bCellLymphNodeSavedPoints = (float*)malloc(model.numPointsLN * sizeof(float));
    model.plasmaCellLymphNodeSavedPoints = (float*)malloc(model.numPointsLN * sizeof(float));

    model.dendriticLymphNode = (float*)malloc(2 * sizeof(float));
    model.tCytotoxicLymphNode = (float*)malloc(2 * sizeof(float));
    model.tHelperLymphNode = (float*)malloc(2 * sizeof(float));
    model.antibodyLymphNode = (float*)malloc(2 * sizeof(float));
    model.bCellLymphNode = (float*)malloc(2 * sizeof(float));
    model.plasmaCellLymphNode = (float*)malloc(2 * sizeof(float));    

    float dendriticLN = 0.0, thelperLN = 0.0, tcytotoxicLN = 0.0, bcellLN = 0.0, plasmacellLN = 0.0, antibodyLN = 0.0;
    InitialConditionLymphNode(&model, dendriticLN, thelperLN, tcytotoxicLN, bcellLN, plasmacellLN, antibodyLN);
    InitialConditionTissueMicroglia(&model);
    return model;
}

float* EquationsLymphNode(structModel model, float* populationLN, int stepPos){
    float* result =(float *)malloc(sizeof(float)*6);
    
    float dcLN = populationLN[0];
    float tCytoLN = populationLN[1];
    float tHelperLN = populationLN[2];
    float bCellLN = populationLN[3];
    float plasmaCellLN = populationLN[4];
    float antibodyLN = populationLN[5];

    //Describe equations

    //Dendritic cell
    float activatedDcMigration = model.parametersModel.gammaD * (model.activatedDCTissueVessels - dcLN) * (float)(model.parametersModel.V_PV/model.parametersModel.V_LN);
    float activatedDcClearance = model.parametersModel.cDl * dcLN;
    result[0] = activatedDcMigration - activatedDcClearance;

    //T Cytotoxic
    float tCytoActivation = model.parametersModel.bTCytotoxic * (model.parametersModel.rhoTCytotoxic*tCytoLN*dcLN - tCytoLN*dcLN);
    float tCytoHomeostasis = model.parametersModel.alphaTCytotoxic * (model.parametersModel.estableTCytotoxic - tCytoLN);
    float tCytoMigration = model.parametersModel.gammaT * (tCytoLN - model.tCytotoxicTissueVessels) * (float)(model.parametersModel.V_BV/model.parametersModel.V_LN);
    result[1] = tCytoActivation + tCytoHomeostasis - tCytoMigration;

    //T Helper
    float tHelperActivation = model.parametersModel.bTHelper * (model.parametersModel.rhoTHelper * tHelperLN * dcLN - tHelperLN * dcLN);
    float tHelperHomeostasis = model.parametersModel.alphaTHelper * (model.parametersModel.estableTHelper - tHelperLN);
    float tHelperDispendure = model.parametersModel.bRho * dcLN * tHelperLN * bCellLN;
    result[2] = tHelperActivation + tHelperHomeostasis - tHelperDispendure;

    //B Cell
    float bCellActivation = model.parametersModel.bRhoB * (model.parametersModel.rhoB * tHelperLN * dcLN - tHelperLN * dcLN * bCellLN);
    float bcellHomeostasis = model.parametersModel.alphaB * (model.parametersModel.estableB - bCellLN);
    result[3] = bcellHomeostasis + bCellActivation;

    //Plasma Cells
    float plasmaActivation = model.parametersModel.bRhoP * (model.parametersModel.rhoP * tHelperLN * dcLN * bCellLN);
    float plasmaHomeostasis = model.parametersModel.alphaP * (model.parametersModel.estableP - plasmaCellLN);
    result[4] = plasmaHomeostasis + plasmaActivation;

    //Antibody
    float antibodyProduction = model.parametersModel.rhoAntibody * plasmaCellLN;
    float antibodyDecayment = model.parametersModel.cF * antibodyLN;
    float antibodyMigration = model.parametersModel.gammaAntibody * (antibodyLN - model.antibodyTissueVessels) * (float)(model.parametersModel.V_BV/model.parametersModel.V_LN);
    result[5] = antibodyProduction - antibodyMigration;

    return result;
}

void verifyValues(structModel model, float value, int time, char* populationName){
    if(value < 0 ||  isnanf(value)){
        printf("Error: %s = (%f) :: time = %f\n", populationName, value, time*model.ht);
        exit(0);
    }
}

void SolverLymphNode(structModel *model, int stepPos){
    float populationLN[6];
    int stepKMinus = stepPos%2;
    int stepKPlus = (stepKMinus+1)%2;
    populationLN[0] = model->dendriticLymphNode[stepKMinus];
    populationLN[1] = model->tCytotoxicLymphNode[stepKMinus];
    populationLN[2] = model->tHelperLymphNode[stepKMinus];
    populationLN[3] = model->bCellLymphNode[stepKMinus];
    populationLN[4] = model->plasmaCellLymphNode[stepKMinus];
    populationLN[5] = model->antibodyLymphNode[stepKMinus];
    
    float* solutionLN;
    solutionLN = EquationsLymphNode(*model, populationLN, stepPos);
    
    //Execute Euler 
    model->dendriticLymphNode[stepKPlus] = model->dendriticLymphNode[stepKMinus] + model->ht*solutionLN[0];
    model->tCytotoxicLymphNode[stepKPlus] = model->tCytotoxicLymphNode[stepKMinus] + model->ht*solutionLN[1];
    model->tHelperLymphNode[stepKPlus] = model->tHelperLymphNode[stepKMinus] + model->ht*solutionLN[2];
    model->bCellLymphNode[stepKPlus] = model->bCellLymphNode[stepKMinus] + model->ht*solutionLN[3];
    model->plasmaCellLymphNode[stepKPlus] = model->plasmaCellLymphNode[stepKMinus] + model->ht*solutionLN[4];
    model->antibodyLymphNode[stepKPlus] = model->antibodyLymphNode[stepKMinus] + model->ht*solutionLN[5];
    free(solutionLN);

    int intervalPoints = (int)(model->tSize/model->numPointsLN);
    if(stepPos%intervalPoints){
        int posSave = stepPos/intervalPoints;
        model->dendriticLymphNodeSavedPoints[posSave] = model->dendriticLymphNode[stepKPlus];
        model->tCytotoxicLymphNodeSavedPoints[posSave] = model->tCytotoxicLymphNode[stepKPlus];
        model->tHelperLymphNodeSavedPoints[posSave] = model->tHelperLymphNode[stepKPlus];
        model->bCellLymphNodeSavedPoints[posSave] = model->bCellLymphNode[stepKPlus];
        model->plasmaCellLymphNodeSavedPoints[posSave] = model->plasmaCellLymphNode[stepKPlus];
        model->antibodyLymphNodeSavedPoints[posSave] = model->antibodyLymphNode[stepKPlus];
    }
    verifyValues(*model, model->dendriticLymphNode[stepKPlus], stepPos, "DC lymph node");
    verifyValues(*model, model->tCytotoxicLymphNode[stepKPlus], stepPos, "CD8 T lymph node");
    verifyValues(*model, model->tHelperLymphNode[stepKPlus], stepPos, "CD4 T lymph node");
    verifyValues(*model, model->bCellLymphNode[stepKPlus], stepPos, "B cell lymph node");
    verifyValues(*model, model->plasmaCellLymphNode[stepKPlus], stepPos, "Plasma cell lymph node");
    verifyValues(*model, model->antibodyLymphNode[stepKPlus], stepPos, "Antibody lymph node");
}

__device__ float upperNeumannBC = 0.0, lowerNeumannBC = 0.0, leftNeumannBC = 0.0, rightNeumannBC = 0.0, hx, ht;
__device__ int xSize;
const int threadsPerBlock = 256;
const int numBlocks = 64;

__global__ void kernelPDE(structParameters* devParams, int kTime, int xSize, float* tCytoSumVessel, float* activatedDCSumVessel, float* antibodySumVessel, float *devActivatedDCLymphNode, float *devAntibodyLymphNode, float *devTCytotoxicLymphNode, float *devThetaPV, float *devThetaBV, float *devMicrogliaKMinus, float *devMicrogliaKPlus, float *devTCytotoxicKMinus, float *devTCytotoxicKPlus, float *devAntibodyKMinus, float *devAntibodyKPlus, float *devConventionalDCKMinus, float *devConventionalDCKPlus, float *devActivatedDCKMinus, float *devActivatedDCKPlus, float *devOligodendrocyteKMinus, float *devOligodendrocyteKPlus){
    int thrIdx = blockIdx.x*blockDim.x + threadIdx.x;
    int vesselIdx = threadIdx.x;
    int line = (int)thrIdx/xSize;
    int column = thrIdx%xSize;

    __shared__ float tCytoSumVesselBlock[threadsPerBlock];
    __shared__ float conventionalDCSumVesselBlock[threadsPerBlock];
    __shared__ float antibodySumVesselBlock[threadsPerBlock];
    while(thrIdx < xSize*xSize){
        line = (int)thrIdx/xSize;
        column = thrIdx%xSize;

        //Define gradient ODCs
        float valIPlus = (line != xSize-1)? devOligodendrocyteKMinus[thrIdx + xSize]: devOligodendrocyteKMinus[thrIdx];
        float valJPlus = (column != xSize-1)? devOligodendrocyteKMinus[thrIdx + 1]: devOligodendrocyteKMinus[thrIdx];
        float valIMinus = (line != 0)? devOligodendrocyteKMinus[thrIdx - xSize]: devOligodendrocyteKMinus[thrIdx];
        float valJMinus = (column != 0)? devOligodendrocyteKMinus[thrIdx - 1]: devOligodendrocyteKMinus[thrIdx];
        
        float gradientOdcI = (float)(valIPlus - valIMinus)/(float)(2*hx);
        float gradientOdcJ = (float)(valJPlus - valJMinus)/(float)(2*hx);

        //Diffusion and Chemotaxis Mic

        valIPlus  = (line != xSize-1)? devMicrogliaKMinus[thrIdx + xSize]: devMicrogliaKMinus[thrIdx] - (float)(2*hx*lowerNeumannBC);
        valJPlus  = (column != xSize-1)? devMicrogliaKMinus[thrIdx + 1]: devMicrogliaKMinus[thrIdx] - (float)(2*hx*rightNeumannBC);
        valIMinus = (line != 0)? devMicrogliaKMinus[thrIdx - xSize]: devMicrogliaKMinus[thrIdx] - (float)(2*hx*upperNeumannBC);
        valJMinus = (column != 0)? devMicrogliaKMinus[thrIdx - 1]: devMicrogliaKMinus[thrIdx] - (float)(2*hx*leftNeumannBC);
        
        float microgliaDiffusion = devParams->micDiffusion*CalculateDiffusion(hx, valJPlus, valJMinus, valIPlus, valIMinus, devMicrogliaKMinus[thrIdx]);
        float microgliaChemotaxis = devParams->chi*CalculateChemottaxis(hx, valJPlus, valJMinus, valIPlus, valIMinus, devMicrogliaKMinus[thrIdx],\
        devParams->avgMic, gradientOdcI, gradientOdcJ);

        //Diffusion and Chemotaxis CDC

        valIPlus  = (line != xSize-1)?devConventionalDCKMinus[thrIdx + xSize]:devConventionalDCKMinus[thrIdx] - (float)(2*hx*lowerNeumannBC);
        valJPlus  = (column != xSize-1)?devConventionalDCKMinus[thrIdx + 1]:devConventionalDCKMinus[thrIdx] - (float)(2*hx*rightNeumannBC);
        valIMinus = (line != 0)?devConventionalDCKMinus[thrIdx - xSize]:devConventionalDCKMinus[thrIdx] - (float)(2*hx*upperNeumannBC);
        valJMinus = (column != 0)?devConventionalDCKMinus[thrIdx - 1]:devConventionalDCKMinus[thrIdx] - (float)(2*hx*leftNeumannBC);

        float conventionalDcDiffusion = devParams->cDcDiffusion*CalculateDiffusion(hx, valJPlus, valJMinus, valIPlus, valIMinus,devConventionalDCKMinus[thrIdx]);
        float conventionalDcChemotaxis = devParams->chi*CalculateChemottaxis(hx, valJPlus, valJMinus, valIPlus, valIMinus,devConventionalDCKMinus[thrIdx],\
        devParams->avgDc, gradientOdcI, gradientOdcJ);

        //Difussion and Chemotaxis CD8T

        valIPlus  = (line != xSize-1)? devTCytotoxicKMinus[thrIdx + xSize]: devTCytotoxicKMinus[thrIdx] - (float)(2*hx*lowerNeumannBC);
        valJPlus  = (column != xSize-1)? devTCytotoxicKMinus[thrIdx + 1]: devTCytotoxicKMinus[thrIdx] - (float)(2*hx*rightNeumannBC);
        valIMinus = (line != 0)? devTCytotoxicKMinus[thrIdx - xSize]: devTCytotoxicKMinus[thrIdx] - (float)(2*hx*upperNeumannBC);
        valJMinus = (column != 0)? devTCytotoxicKMinus[thrIdx - 1]: devTCytotoxicKMinus[thrIdx] - (float)(2*hx*leftNeumannBC);

        float tCytotoxicDiffusion = devParams->tCytoDiffusion*CalculateDiffusion(hx, valJPlus, valJMinus, valIPlus, valIMinus, devTCytotoxicKMinus[thrIdx]);
        float tCytotoxicChemotaxis = devParams->chi*CalculateChemottaxis(hx, valJPlus, valJMinus, valIPlus, valIMinus, devTCytotoxicKMinus[thrIdx],\
        devParams->avgT, gradientOdcI, gradientOdcJ);

        //Difussion ADC

        valIPlus  = (line != xSize-1)? devActivatedDCKMinus[thrIdx + xSize]: devActivatedDCKMinus[thrIdx] - (float)(2*hx*lowerNeumannBC);
        valJPlus  = (column != xSize-1)? devActivatedDCKMinus[thrIdx + 1]: devActivatedDCKMinus[thrIdx] - (float)(2*hx*rightNeumannBC);
        valIMinus = (line != 0)? devActivatedDCKMinus[thrIdx - xSize]: devActivatedDCKMinus[thrIdx] - (float)(2*hx*upperNeumannBC);
        valJMinus = (column != 0)? devActivatedDCKMinus[thrIdx - 1]: devActivatedDCKMinus[thrIdx] - (float)(2*hx*leftNeumannBC);

        float activatedDCDiffusion = devParams->aDcDiffusion*CalculateDiffusion(hx, valJPlus, valJMinus, valIPlus, valIMinus, devActivatedDCKMinus[thrIdx]);

        //Difussion Antibody

        valIPlus  = (line != xSize-1)? devAntibodyKMinus[thrIdx + xSize]: devAntibodyKMinus[thrIdx] - (float)(2*hx*lowerNeumannBC);
        valJPlus  = (column != xSize-1)? devAntibodyKMinus[thrIdx + 1]: devAntibodyKMinus[thrIdx] - (float)(2*hx*rightNeumannBC);
        valIMinus = (line != 0)? devAntibodyKMinus[thrIdx - xSize]: devAntibodyKMinus[thrIdx] - (float)(2*hx*upperNeumannBC);
        valJMinus = (column != 0)? devAntibodyKMinus[thrIdx - 1]: devAntibodyKMinus[thrIdx] - (float)(2*hx*leftNeumannBC);

        float antibodyDiffusion = devParams->antibodyDiffusion*CalculateDiffusion(hx, valJPlus, valJMinus, valIPlus, valIMinus, devAntibodyKMinus[thrIdx]);

        //*******************************************Solving Tissue equations*****************************************************

        //Microglia update
        float microgliaReaction = devParams->muMic*devMicrogliaKMinus[thrIdx]*(devParams->avgMic - devMicrogliaKMinus[thrIdx]);
        float microgliaClearance = devParams->cMic*devMicrogliaKMinus[thrIdx];

        devMicrogliaKPlus[thrIdx] = devMicrogliaKMinus[thrIdx] + \
        ht*(microgliaDiffusion - microgliaChemotaxis + microgliaReaction - microgliaClearance);
        if((devMicrogliaKPlus[thrIdx]) < 0 || isnanf (devMicrogliaKPlus[thrIdx])){
            printf("Microglia (%f) deu erro no tempo %f\n", devMicrogliaKPlus[thrIdx], kTime*ht);
            exit(0);
        }

        //Conventional DC update
        float conventionalDcReaction = devParams->muCDc*devOligodendrocyteKMinus[thrIdx]*(devParams->avgDc - devConventionalDCKMinus[thrIdx]);
        float conventionalDcActivation = devParams->bD*devConventionalDCKMinus[thrIdx]*devOligodendrocyteKMinus[thrIdx];
        float conventionalDcClearance = devParams->cCDc*devConventionalDCKMinus[thrIdx];

        devConventionalDCKPlus[thrIdx] = devConventionalDCKMinus[thrIdx] + \
        ht*(conventionalDcDiffusion - conventionalDcChemotaxis - conventionalDcClearance + conventionalDcReaction - conventionalDcActivation);
        if((devConventionalDCKPlus[thrIdx]) < 0 || isnanf (devConventionalDCKPlus[thrIdx])){
            printf("CDC (%f) deu erro no tempo %f\n", devConventionalDCKPlus[thrIdx], kTime*ht);
            exit(0);
        }

        //Activated DC update
        float activatedDcClearance = devParams->cADc*devActivatedDCKMinus[thrIdx];
        float activatedDcMigration = devThetaPV[thrIdx]*devParams->gammaD*(*devActivatedDCLymphNode - devActivatedDCKMinus[thrIdx]);
        
        devActivatedDCKPlus[thrIdx] = devActivatedDCKMinus[thrIdx] + ht*(activatedDCDiffusion + conventionalDcActivation + activatedDcMigration - activatedDcClearance);
        if((devActivatedDCKPlus[thrIdx]) < 0 || isnanf (devActivatedDCKPlus[thrIdx])){
            printf("ADC (%f) deu erro no tempo %f\n", devActivatedDCKPlus[thrIdx], kTime*ht);
            exit(0);
        }

        //CD8 T update
        float tCytotoxicMigration = devThetaBV[thrIdx]*devParams->gammaT*(*devTCytotoxicLymphNode - devTCytotoxicKMinus[thrIdx]);
        
        devTCytotoxicKPlus[thrIdx] = devTCytotoxicKMinus[thrIdx] + ht*(tCytotoxicDiffusion - tCytotoxicChemotaxis + tCytotoxicMigration);
        if((devTCytotoxicKPlus[thrIdx]) < 0 || isnanf (devTCytotoxicKPlus[thrIdx])){
            printf("tCytotoxic (%f) deu erro no tempo %f\n", devTCytotoxicKPlus[thrIdx], kTime*ht);
            exit(0);
        }

        //Antibody update
        float odcAntibodyMicrogliaFagocitosis = devParams->lambAntMic*devAntibodyKMinus[thrIdx]*(devParams->avgOdc - devOligodendrocyteKMinus[thrIdx])*fFunc(devMicrogliaKMinus[thrIdx], devParams->avgMic);
        float antibodyMigration = devThetaBV[thrIdx]*devParams->gammaAntibody*(*devAntibodyLymphNode - devAntibodyKMinus[thrIdx]);
        
        devAntibodyKPlus[thrIdx] = devAntibodyKMinus[thrIdx] + ht*(antibodyDiffusion + antibodyMigration - odcAntibodyMicrogliaFagocitosis);
        if((devAntibodyKPlus[thrIdx]) < 0 || isnanf (devAntibodyKPlus[thrIdx])){
            printf("antibody (%.8f) deu erro no tempo %f\n", (devAntibodyKPlus[thrIdx]), kTime*ht);
            exit(0);
        }

        //Oligodendrocytes update
        float odcMicrogliaFagocitosis = devParams->rM*fFunc(devMicrogliaKMinus[thrIdx], devParams->avgMic)*(devParams->avgOdc - devOligodendrocyteKMinus[thrIdx]);
        float odcTCytotoxicApoptosis = devParams->rT*fFunc(devTCytotoxicKMinus[thrIdx], devParams->avgT)*(devParams->avgOdc - devOligodendrocyteKMinus[thrIdx]);

        devOligodendrocyteKPlus[thrIdx] = devOligodendrocyteKMinus[thrIdx] + ht*(odcAntibodyMicrogliaFagocitosis + odcMicrogliaFagocitosis + odcTCytotoxicApoptosis);
        if((devOligodendrocyteKPlus[thrIdx]) < 0 || isnanf (devOligodendrocyteKPlus[thrIdx])){
            printf("oligodendrocyte (%f) deu erro no tempo %f\n", devOligodendrocyteKPlus[thrIdx], kTime*ht);
            exit(0);
        }
        if(devThetaBV[thrIdx] == 1){
            tCytoSumVesselBlock[vesselIdx] += devTCytotoxicKPlus[thrIdx];
            antibodySumVesselBlock[vesselIdx] += devAntibodyKPlus[thrIdx];
        }
        if(devThetaPV[thrIdx] == 1){
            conventionalDCSumVesselBlock[vesselIdx] += devActivatedDCKPlus[thrIdx];
        }
        thrIdx += gridDim.x*blockDim.x;
    }
    __syncthreads();
    int i = blockDim.x / 2;
    while (i != 0) {
        if (vesselIdx < i){
            tCytoSumVesselBlock[vesselIdx] += tCytoSumVesselBlock[vesselIdx + i];
            conventionalDCSumVesselBlock[vesselIdx] += conventionalDCSumVesselBlock[vesselIdx + i];
            antibodySumVesselBlock[vesselIdx] += antibodySumVesselBlock[vesselIdx + i];
        }
        __syncthreads();
        i /= 2;
    }
    if (vesselIdx == 0){
        tCytoSumVessel[blockIdx.x] = tCytoSumVesselBlock[0];
        activatedDCSumVessel[blockIdx.x] = conventionalDCSumVesselBlock[0];
        antibodySumVessel[blockIdx.x] = antibodySumVesselBlock[0];
    }

}

void RunModel(structModel *model){
    //Save IC
    WriteFiles(*model, model->oligodendrocyte[0], model->microglia[0], model->tCytotoxic[0], model->antibody[0], model->conventionalDc[0], model->activatedDc[0], 0);
    
    float sumActivatedDCLymphNode, sumAntibodyLymphNode, sumTCytotoxicLymphNode;

    float *devThetaPV, *devThetaBV, *devActivatedDCVessel, *devTCytotoxicVessel, *devAntibodyVessel, *devActivatedDCLymphNode, *devAntibodyLymphNode, *devTCytotoxicLymphNode, *devMicrogliaKMinus, *devMicrogliaKPlus, *devTCytotoxicKMinus, *devTCytotoxicKPlus, *devAntibodyKMinus, *devAntibodyKPlus, *devConventionalDCKMinus, *devConventionalDCKPlus, *devActivatedDCKMinus, *devActivatedDCKPlus, *devOligodendrocytesDCKMinus, *devOligodendrocytesDCKPlus;
    
    hipMalloc((void**)&devThetaPV, model->xSize*model->xSize*sizeof(float));
    hipMalloc((void**)&devThetaBV, model->xSize*model->xSize*sizeof(float));

    hipMalloc((void**)&devOligodendrocytesDCKMinus, model->xSize*model->xSize*sizeof(float));
    hipMalloc((void**)&devOligodendrocytesDCKPlus, model->xSize*model->xSize*sizeof(float));

    hipMalloc((void**)&devMicrogliaKMinus, model->xSize*model->xSize*sizeof(float));
    hipMalloc((void**)&devMicrogliaKPlus, model->xSize*model->xSize*sizeof(float));
    
    hipMalloc((void**)&devTCytotoxicKMinus, model->xSize*model->xSize*sizeof(float));
    hipMalloc((void**)&devTCytotoxicKPlus, model->xSize*model->xSize*sizeof(float));
    
    hipMalloc((void**)&devAntibodyKMinus, model->xSize*model->xSize*sizeof(float));
    hipMalloc((void**)&devAntibodyKPlus, model->xSize*model->xSize*sizeof(float));
    
    hipMalloc((void**)&devConventionalDCKMinus, model->xSize*model->xSize*sizeof(float));
    hipMalloc((void**)&devConventionalDCKPlus, model->xSize*model->xSize*sizeof(float));
    
    hipMalloc((void**)&devActivatedDCKMinus, model->xSize*model->xSize*sizeof(float));
    hipMalloc((void**)&devActivatedDCKPlus, model->xSize*model->xSize*sizeof(float));
    
    hipMemcpy(devOligodendrocytesDCKMinus, &model->oligodendrocyte[0], model->xSize*model->xSize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devMicrogliaKMinus, &model->microglia[0], model->xSize*model->xSize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devTCytotoxicKMinus, &model->tCytotoxic[0], model->xSize*model->xSize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devAntibodyKMinus, &model->antibody[0], model->xSize*model->xSize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devConventionalDCKMinus, &model->conventionalDc[0], model->xSize*model->xSize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devActivatedDCKMinus, &model->activatedDc[0], model->xSize*model->xSize*sizeof(float), hipMemcpyHostToDevice);
    
    structParameters* devParams;
    printf("tamanho parametros = %d", sizeof(devParams));
    //se der errado passar parametro por parametro (tentar com memoria de constantes)
    hipMalloc((void**)&devParams, sizeof(structParameters));
    hipMemcpy(devParams, &model->parametersModel, sizeof(structParameters), hipMemcpyHostToDevice);

    hipMalloc((void**)&devActivatedDCLymphNode, sizeof(float));
    hipMalloc((void**)&devAntibodyLymphNode, sizeof(float));
    hipMalloc((void**)&devTCytotoxicLymphNode, sizeof(float));

    hipMalloc((void**)&devActivatedDCVessel, numBlocks*sizeof(float));
    hipMalloc((void**)&devAntibodyVessel, numBlocks*sizeof(float));
    hipMalloc((void**)&devTCytotoxicVessel, numBlocks*sizeof(float));
    //Inicializar os constant com os valores
    int stepKMinus = 0, stepKPlus;
    
    float valIPlus = 0.0, valIMinus = 0.0, valJPlus = 0.0, valJMinus = 0.0, gradientOdcI = 0.0, gradientOdcJ = 0.0;

    float microgliaChemotaxis = 0.0, tCytotoxicChemotaxis = 0.0, conventionalDcChemotaxis = 0.0,\
     microgliaDiffusion = 0.0, tCytotoxicDiffusion = 0.0, conventionalDcDiffusion = 0.0, activatedDCDiffusion = 0.0, antibodyDiffusion = 0.0;

    float microgliaReaction = 0.0, microgliaClearance = 0.0, tCytotoxicMigration = 0.0, odcAntibodyMicrogliaFagocitosis = 0.0, \
    odcMicrogliaFagocitosis = 0.0, odcTCytotoxicApoptosis = 0.0, conventionalDcReaction = 0.0, conventionalDcClearance = 0.0, conventionalDcActivation = 0.0, \
    activatedDcClearance = 0.0, activatedDcMigration = 0.0, antibodyMigration = 0.0;

    float microgliaKMinus = 0.0, conventionalDcKMinus = 0.0, activatedDcKMinus = 0.0, tCytotoxicKMinus = 0.0, antibodyKMinus = 0.0, oligodendrocyteKMinus = 0.0;

    float auxAdcPV = 0.0, auxAntibodyBV = 0.0, auxTCytotoxicBV = 0.0;

    for(int kTime = 1; kTime <= model->tSize; kTime++){
        auxAdcPV = 0.0, auxAntibodyBV = 0.0, auxTCytotoxicBV = 0.0;
        // solve lymphnode
        SolverLymphNode(model, kTime);

        //copiar LN pra GPU
        hipMemcpy(devActivatedDCLymphNode, &model->dendriticLymphNode[stepKPlus], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(devAntibodyLymphNode, &model->antibodyLymphNode[stepKPlus], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(devTCytotoxicLymphNode, &model->tCytotoxicLymphNode[stepKPlus], sizeof(float), hipMemcpyHostToDevice);        
        stepKPlus = kTime%2;

        if(stepKPlus%2 == 1)
            kernelPDE<<<numBlocks,threadsPerBlock>>>(devParams, kTime, xSize, devTCytotoxicVessel, devActivatedDCVessel, devAntibodyVessel, devActivatedDCLymphNode, devAntibodyLymphNode, devTCytotoxicLymphNode, devThetaPV, devThetaBV, devMicrogliaKMinus, devMicrogliaKPlus, devTCytotoxicKMinus, devTCytotoxicKPlus, devAntibodyKMinus, devAntibodyKPlus, devConventionalDCKMinus, devConventionalDCKPlus, devActivatedDCKMinus, devActivatedDCKPlus, devOligodendrocytesDCKMinus, devOligodendrocytesDCKPlus);
        else
            kernelPDE<<<numBlocks,threadsPerBlock>>>(devParams, kTime, xSize, devTCytotoxicVessel, devActivatedDCVessel, devAntibodyVessel, devActivatedDCLymphNode, devAntibodyLymphNode, devTCytotoxicLymphNode, devThetaPV, devThetaBV, devMicrogliaKPlus, devMicrogliaKMinus, devTCytotoxicKPlus, devTCytotoxicKMinus, devAntibodyKPlus, devAntibodyKMinus, devConventionalDCKPlus, devConventionalDCKMinus, devActivatedDCKPlus, devActivatedDCKMinus, devOligodendrocytesDCKPlus, devOligodendrocytesDCKMinus);
        if(kTime%model->intervalFigures == 0 || kTime == model->tSize){
            //Copia tecido para a CPU
            WriteFiles(*model, model->oligodendrocyte[stepKPlus], model->microglia[stepKPlus], model->tCytotoxic[stepKPlus], model->antibody[stepKPlus], model->conventionalDc[stepKPlus], model->activatedDc[stepKPlus], kTime);
        }
        //Copia do device para o host as integrais do tecido
        model->tCytotoxicTissueVessels = auxTCytotoxicBV/model->parametersModel.V_BV;
        model->antibodyTissueVessels = auxAntibodyBV/model->parametersModel.V_BV;
        model->activatedDCTissueVessels = auxAdcPV/model->parametersModel.V_PV;
        stepKMinus += 1;
        stepKMinus = stepKMinus%2;
    }
    printf("Computation Done!!\n");
    // printf("Saving results...\n\n");
    // WriteLymphNodeFiles(*model, model->dendriticLymphNodeSavedPoints, model->tHelperLymphNodeSavedPoints, model->tCytotoxicLymphNodeSavedPoints, model->bCellLymphNodeSavedPoints, model->plasmaCellLymphNodeSavedPoints, model->antibodyLymphNodeSavedPoints);
    // PlotResults(*model);
}